#include "hip/hip_runtime.h"
#include "../NativeBlas.h"
#include <hipblas.h>
#include <hipsolver.h>
#include <pointercast.h>
#include <stdio.h>
#include <types/float16.h>



hipblasStatus_t convertStatus(int status) {
    switch (status) {
        case 0:
            return HIPBLAS_STATUS_SUCCESS;
        case 1:
            return HIPBLAS_STATUS_NOT_INITIALIZED;
        case 3:
            return HIPBLAS_STATUS_ALLOC_FAILED;
        case 7:
            return HIPBLAS_STATUS_INVALID_VALUE;
        case 8:
            return HIPBLAS_STATUS_ARCH_MISMATCH;
        case 11:
            return HIPBLAS_STATUS_MAPPING_ERROR;
        case 13:
            return HIPBLAS_STATUS_EXECUTION_FAILED;
        case 14:
            return HIPBLAS_STATUS_INTERNAL_ERROR;
        case 15:
            return HIPBLAS_STATUS_NOT_SUPPORTED;
        case 16:
            return HIPBLAS_STATUS_UNKNOWN;
        default:
            return HIPBLAS_STATUS_SUCCESS;
    }
}

hipblasFillMode_t convertUplo(int fillMode) {
    switch (fillMode) {
        case 0:
            return HIPBLAS_FILL_MODE_LOWER;
        case 1:
            return HIPBLAS_FILL_MODE_UPPER;
        default:
            return HIPBLAS_FILL_MODE_LOWER;
    }
}

hipblasDiagType_t convertDiag(int diag) {
    switch (diag) {
        case 0:
            return HIPBLAS_DIAG_NON_UNIT;
        case 1:
            return HIPBLAS_DIAG_UNIT;
        default:
            return HIPBLAS_DIAG_NON_UNIT;
    }
}

hipblasOperation_t convertTranspose(int op) {
    switch(op) {
        case 78: return HIPBLAS_OP_N;
        case 84: return HIPBLAS_OP_T;
        case 67: return HIPBLAS_OP_C;
        default: return HIPBLAS_OP_N;
    }
}

hipblasPointerMode_t convertPointerMode(int pointerMode) {
    switch(pointerMode) {
        case 0: return HIPBLAS_POINTER_MODE_HOST;
        case 1: return HIPBLAS_POINTER_MODE_DEVICE;
        default: return HIPBLAS_POINTER_MODE_HOST;
    }}

hipblasSideMode_t convertSideMode(int sideMode) {
    switch(sideMode) {
        case 0: return HIPBLAS_SIDE_LEFT;
        case 1: return HIPBLAS_SIDE_RIGHT;
        default: return HIPBLAS_SIDE_LEFT;
    }
}

void Nd4jBlas::setMaxThreads(int num) {
    // no-op
}

int Nd4jBlas::getMaxThreads() {
    // 0 - cuBLAS
    return 0;
}

int Nd4jBlas::getVendor() {
    // 1 - cuBLAS
    return 1;
}


/*
 * ======================================================
 * Level 1 BLAS functions
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * DOT
 * ------------------------------------------------------
 */

float Nd4jBlas::sdsdot(Nd4jPointer *extraParams, int N, float alpha,
                       float *X, int incX,
                       float *Y, int incY) {
//    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    //nothing found?
    return 0.0f;
}

double Nd4jBlas::dsdot(Nd4jPointer *extraParams, int N,
                       float *X, int incX,
                       float *Y, int incY) {
//    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    //nothing found?
    return 0.0f;
}

double Nd4jBlas::ddot(Nd4jPointer *extraParams, int N,
                      double *X, int incX,
                      double *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    double resultPointer = 0.0f;
    hipblasDdot(*handle, N, X, incX, Y, incY, &resultPointer);
    return resultPointer;
}

float Nd4jBlas::sdot(Nd4jPointer *extraParams, int N,
                     float *X, int incX,
                     float *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    float resultPointer = 0.0f;
    hipblasSdot(*handle, N, X, incX, Y, incY, &resultPointer);
    return resultPointer;
}

/*
 * ------------------------------------------------------
 * NRM2
 * ------------------------------------------------------
 */

float Nd4jBlas::snrm2(Nd4jPointer *extraParams, int N, float *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    float resultPointer = 0.0f;
    hipblasSnrm2(*handle, N, X, incX, &resultPointer);
    return resultPointer;


}

double Nd4jBlas::dnrm2(Nd4jPointer *extraParams, int N, double *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    double resultPointer = 0.0;
    hipblasDnrm2(*handle, N, X, incX, &resultPointer);
    return resultPointer;
}

/*
 * ------------------------------------------------------
 * ASUM
 * ------------------------------------------------------
 */

float Nd4jBlas::sasum(Nd4jPointer *extraParams, int N, float *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    float resultPointer = 0.0f;
    hipblasSasum(*handle, N, X, incX, &resultPointer);
    return resultPointer;


}

double Nd4jBlas::dasum(Nd4jPointer *extraParams, int N, double *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    double resultPointer = 0.0f;
    hipblasDasum(*handle, N, X, incX, &resultPointer);
    return resultPointer;

}

/*
 * ------------------------------------------------------
 * IAMAX
 * ------------------------------------------------------
 */

int Nd4jBlas::isamax(Nd4jPointer *extraParams, int N, float *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    int resultPointer = 0;
    hipblasIsamax(*handle,N,X,incX,&resultPointer);
    return resultPointer;

}

int Nd4jBlas::idamax(Nd4jPointer *extraParams, int N, double *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    int resultPointer = 0;
    hipblasIdamax(*handle, N, X, incX, &resultPointer);
    return resultPointer;


}

/*
 * ======================================================
 * Level 1 BLAS procedures
 * ======================================================
 */

/*
 * ------------------------------------------------------
 * ROT
 * ------------------------------------------------------
 */

void Nd4jBlas::srot(Nd4jPointer *extraParams, int N,
                    float *X, int incX,
                    float *Y, int incY,
                    float c, float s) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSrot(*handle, N, X, incX, Y, incY, &c, &s);
}

void Nd4jBlas::drot(Nd4jPointer *extraParams, int N,
                    double *X, int incX,
                    double *Y, int incY,
                    double c, double s) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDrot(*handle, N, X, incX, Y, incY, &c, &s);
}

/*
 * ------------------------------------------------------
 * ROTG
 * ------------------------------------------------------
 */

void Nd4jBlas::srotg(Nd4jPointer *extraParams, float *args) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSrotg(*handle, &args[0], &args[1], &args[2], &args[3]);
}

void Nd4jBlas::drotg(Nd4jPointer *extraParams, double *args) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDrotg(*handle, &args[0], &args[1], &args[2], &args[3]);

}

/*
 * ------------------------------------------------------
 * ROTMG
 * ------------------------------------------------------
 */

void Nd4jBlas::srotmg(Nd4jPointer *extraParams, float *args,
                      float *P) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSrotmg(*handle, &args[0], &args[1], &args[2], &args[3], P);

}

void Nd4jBlas::drotmg(Nd4jPointer *extraParams, double *args,
                      double *P) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDrotmg(*handle, &args[0], &args[1], &args[2], &args[3], P);

}

/*
 * ------------------------------------------------------
 * ROTM
 * ------------------------------------------------------
 */

void Nd4jBlas::srotm(Nd4jPointer *extraParams, int N,
                     float *X, int incX,
                     float *Y, int incY,
                     float *P) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSrotm(*handle, N, X, incX, Y, incY, P);

}

void Nd4jBlas::drotm(Nd4jPointer *extraParams, int N,
                     double *X, int incX,
                     double *Y, int incY,
                     double *P) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDrotm(*handle, N, X, incX, Y, incY, P);

}

/*
 * ------------------------------------------------------
 * SWAP
 * ------------------------------------------------------
 */

void Nd4jBlas::sswap(Nd4jPointer *extraParams, int N,
                     float *X, int incX,
                     float *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSswap(*handle, N, X, incX, Y, incY);
}

void Nd4jBlas::dswap(Nd4jPointer *extraParams, int N,
                     double *X, int incX,
                     double *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDswap(*handle, N, X, incX, Y, incY);

}

/*
 * ------------------------------------------------------
 * SCAL
 * ------------------------------------------------------
 */

void Nd4jBlas::sscal(Nd4jPointer *extraParams, int N, float alpha,
                     float *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSscal(*handle, N, &alpha, X, incX);

}

void Nd4jBlas::dscal(Nd4jPointer *extraParams, int N, double alpha,
                     double *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDscal(*handle, N, &alpha, X, incX);
}

/*
 * ------------------------------------------------------
 * SCOPY
 * ------------------------------------------------------
 */

void Nd4jBlas::scopy(Nd4jPointer *extraParams, int N,
                     float *X, int incX,
                     float *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasScopy(*handle, N, X, incX, Y, incY);
}

void Nd4jBlas::dcopy(Nd4jPointer *extraParams, int N,
                     double *X, int incX,
                     double *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDcopy(*handle, N, X, incX, Y, incY);

}

/*
 * ------------------------------------------------------
 * AXPY
 * ------------------------------------------------------
 */

void Nd4jBlas::saxpy(Nd4jPointer *extraParams, int N, float alpha,
                     float *X, int incX,
                     float *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSaxpy(*handle, N, &alpha, X, incX, Y, incY);
}

void Nd4jBlas::daxpy(Nd4jPointer *extraParams, int N, double alpha,
                     double *X, int incX,
                     double *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDaxpy(*handle, N, &alpha, X, incX, Y, incY);

}

/*
 * ======================================================
 * Level 2 BLAS procedures
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * GEMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sgemv(Nd4jPointer *extraParams, int Order, int TransA,
                     int M, int N,
                     float alpha,
                     float *A, int lda,
                     float *X, int incX,
                     float beta,
                     float *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSgemv(*handle, convertTranspose(TransA), M, N, &alpha, A, lda, X, incX, &beta, Y,
                   incY);
}

void Nd4jBlas::dgemv(Nd4jPointer *extraParams, int Order, int TransA,
                     int M, int N,
                     double alpha,
                     double *A, int lda,
                     double *X, int incX,
                     double beta,
                     double *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDgemv(*handle,convertTranspose(TransA),M,N,&alpha,A,lda,X,incX,&beta,Y,incY);
}

/*
 * ------------------------------------------------------
 * GBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sgbmv(Nd4jPointer *extraParams, int Order, int TransA,
                     int M, int N,
                     int KL, int KU,
                     float alpha,
                     float *A, int lda,
                     float *X, int incX,
                     float beta,
                     float *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSgbmv(*handle, convertTranspose(TransA), M, N, KL, KU, &alpha, A, lda, X, incX, &beta,
                   Y, incY);
}

void Nd4jBlas::dgbmv(Nd4jPointer *extraParams, int Order, int TransA,
                     int M, int N,
                     int KL, int KU,
                     double alpha,
                     double *A, int lda,
                     double *X, int incX,
                     double beta,
                     double *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDgbmv(*handle, convertTranspose(TransA), M, N, KL, KU, &alpha, A, lda, X, incX, &beta,
                   Y, incY);
}

/*
 * ------------------------------------------------------
 * SYMV
 * ------------------------------------------------------
 */

void Nd4jBlas::ssymv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     float *A, int lda,
                     float *X, int incX,
                     float beta,
                     float *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsymv(*handle, convertUplo(Uplo), N, &alpha, A, lda, X, incX, &beta, Y, incY);
}

void Nd4jBlas::dsymv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     double *A, int lda,
                     double *X, int incX,
                     double beta,
                     double *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsymv(*handle, convertUplo(Uplo), N, &alpha, A, lda, X, incX, &beta, Y, incY);

}

/*
 * ------------------------------------------------------
 * SBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::ssbmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N, int K,
                     float alpha,
                     float *A, int lda,
                     float *X, int incX,
                     float beta,
                     float *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsbmv(*handle, convertUplo(Uplo), N, K, &alpha, A, lda, X, incX, &beta, Y, incY);
}

void Nd4jBlas::dsbmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N, int K,
                     double alpha,
                     double *A, int lda,
                     double *X, int incX,
                     double beta,
                     double *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsbmv(*handle, convertUplo(Uplo), N, K, &alpha, A, lda, X, incX, &beta, Y, incY);

}

/*
 * ------------------------------------------------------
 * SPMV
 * ------------------------------------------------------
 */

void Nd4jBlas::sspmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     float *Ap,
                     float *X, int incX,
                     float beta,
                     float *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSspmv(*handle, convertUplo(Uplo), N, &alpha, Ap, X, incX, &beta, Y, incY);

}

void Nd4jBlas::dspmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     double *Ap,
                     double *X, int incX,
                     double beta,
                     double *Y, int incY) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDspmv(*handle, convertUplo(Uplo), N, &alpha, Ap, X, incX, &beta, Y, incY);


}

/*
 * ------------------------------------------------------
 * TRMV
 * ------------------------------------------------------
 */

void Nd4jBlas::strmv(Nd4jPointer *extraParams, int Order, int Uplo, int TransA,
                     int Diag,
                     int N, float alpha,
                     float *A, int lda,
                     float *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStrmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, A, lda,
                   X, incX);
}

void Nd4jBlas::dtrmv(Nd4jPointer *extraParams, int Order, int Uplo, int TransA,
                     int Diag,
                     int N, double alpha,
                     double *A, int lda,
                     double *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtrmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, A, lda,
                   X, incX);

}

/*
 * ------------------------------------------------------
 * TBMV
 * ------------------------------------------------------
 */

void Nd4jBlas::stbmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     float *A, int lda,
                     float *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStbmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, A, lda,
                   X, incX);
}

void Nd4jBlas::dtbmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     double *A, int lda,
                     double *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtbmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, A, lda,
                   X, incX);

}

/*
 * ------------------------------------------------------
 * TPMV
 * ------------------------------------------------------
 */

void Nd4jBlas::stpmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     float *Ap,
                     float *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStpmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, Ap, X,
                   incX);

}

void Nd4jBlas::dtpmv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     double *Ap,
                     double *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtpmv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, Ap, X,
                   incX);

}

/*
 * ------------------------------------------------------
 * TRSV
 * ------------------------------------------------------
 */

void Nd4jBlas::strsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     float *A, int lda,
                     float *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStrsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, A, lda, X,
                   incX);
}

void Nd4jBlas::dtrsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     double *A, int lda,
                     double *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtrsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, A, lda, X,
                   incX);

}

/*
 * ------------------------------------------------------
 * TBSV
 * ------------------------------------------------------
 */

void Nd4jBlas::stbsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     float *A, int lda,
                     float *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStbsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, A, lda,
                   X, incX);

}

void Nd4jBlas::dtbsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N, int K,
                     double *A, int lda,
                     double *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtbsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, K, A, lda,
                   X, incX);

}

/*
 * ------------------------------------------------------
 * TPSV
 * ------------------------------------------------------
 */

void Nd4jBlas::stpsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     float *Ap,
                     float *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStpsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, Ap, X,
                   incX);
}

void Nd4jBlas::dtpsv(Nd4jPointer *extraParams, int Order, int Uplo,
                     int TransA, int Diag,
                     int N,
                     double *Ap,
                     double *X, int incX) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtpsv(*handle, convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), N, Ap, X,
                   incX);

}

/*
 * ------------------------------------------------------
 * GER
 * ------------------------------------------------------
 */

void Nd4jBlas::sger(Nd4jPointer *extraParams, int Order,
                    int M, int N,
                    float alpha,
                    float *X, int incX,
                    float *Y, int incY,
                    float *A, int lda) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSger(*handle, M, N, &alpha, X, incX, Y, incY, A, lda);

}

void Nd4jBlas::dger(Nd4jPointer *extraParams, int Order,
                    int M, int N,
                    double alpha,
                    double *X, int incX,
                    double *Y, int incY,
                    double *A, int lda) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDger(*handle, M, N, &alpha, X, incX, Y, incY, A, lda);

}

/*
 * ------------------------------------------------------
 * SYR
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr(Nd4jPointer *extraParams, int Order, int Uplo,
                    int N,
                    float alpha,
                    float *X, int incX,
                    float *A, int lda) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsyr(*handle, convertUplo(Uplo), N, &alpha, X, incX, A, lda);
}

void Nd4jBlas::dsyr(Nd4jPointer *extraParams, int Order, int Uplo,
                    int N,
                    double alpha,
                    double *X, int incX,
                    double *A, int lda) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsyr(*handle, convertUplo(Uplo), N, &alpha, X, incX, A, lda);

}

/*
 * ------------------------------------------------------
 * SPR
 * ------------------------------------------------------
 */

void Nd4jBlas::sspr(Nd4jPointer *extraParams, int Order, int Uplo,
                    int N,
                    float alpha,
                    float *X, int incX,
                    float *Ap) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSspr(*handle, convertUplo(Uplo), N, &alpha, X, incX, Ap);
}

void Nd4jBlas::dspr(Nd4jPointer *extraParams, int Order, int Uplo,
                    int N,
                    double alpha,
                    double *X, int incX,
                    double *Ap) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDspr(*handle, convertUplo(Uplo), N, &alpha, X, incX, Ap);

}

/*
 * ------------------------------------------------------
 * SYR2
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr2(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     float *X, int incX,
                     float *Y, int incY,
                     float *A, int lda) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsyr2(*handle, convertUplo(Uplo), N, &alpha, X, incX, Y, incY, A, lda);

}

void Nd4jBlas::dsyr2(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     double *X, int incX,
                     double *Y, int incY,
                     double *A, int lda) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsyr2(*handle, convertUplo(Uplo), N, &alpha, X, incX, Y, incY, A, lda);

}

/*
 * ------------------------------------------------------
 * SPR2
 * ------------------------------------------------------
 */

void Nd4jBlas::sspr2(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     float alpha,
                     float *X, int incX,
                     float *Y, int incY,
                     float *Ap) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSspr2(*handle, convertUplo(Uplo), N, &alpha, X, incX, Y, incY, Ap);
}

void Nd4jBlas::dspr2(Nd4jPointer *extraParams, int Order, int Uplo,
                     int N,
                     double alpha,
                     double *X, int incX,
                     double *Y, int incY,
                     double *Ap) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDspr2(*handle, convertUplo(Uplo), N, &alpha, X, incX, Y, incY, Ap);

}

/*
 * ======================================================
 * Level 3 BLAS procedures
 * ======================================================
 */


/*
 * ------------------------------------------------------
 * GEMM
 * ------------------------------------------------------
 */
int getIntPtr(Nd4jPointer ptrToDeviceId) {
    return (int)(Nd4jIndex)ptrToDeviceId;
}

void Nd4jBlas::hgemm(Nd4jPointer *extraParams, int Order, int TransA, int TransB,
                     int M, int N, int K,
                     float alpha,
                     float16 *A, int lda,
                     float16 *B, int ldb,
                     float beta,
                     float16 *C, int ldc) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    int arch = getIntPtr(extraParams[1]);

#ifdef CUDA_8
    // HIP_R_16F for CUDA 8
    // CUBLAS_DATA_HALF for CUDA 7.5

    // on these selected archs we run with hipblasHgemm
    if (arch == 53 || arch == 60){
        __half *aPointer = reinterpret_cast<__half *>(A);
        __half *bPointer = reinterpret_cast<__half *>(B);
        __half *cPointer = reinterpret_cast<__half *>(C);

        float16 hAlpha = alpha;
        float16 hBeta = beta;

        hipblasHgemm(*handle,
                convertTranspose(TransA), convertTranspose(TransB),
                M, N, K,
                &hAlpha.data,
                aPointer, lda,
                bPointer, ldb,
                &hBeta.data,
                cPointer, ldc);
    } else {
        cublasSgemmEx(*handle,
                   convertTranspose(TransA),
                   convertTranspose(TransB),
                   M, N, K,
                   &alpha,
                   A, HIP_R_16F, lda,
                   B, HIP_R_16F, ldb,
                   &beta,
                   C, HIP_R_16F, ldc);
    }
#else
    cublasSgemmEx(*handle,
                  convertTranspose(TransA),
                  convertTranspose(TransB),
                  M, N, K,
                  &alpha,
                  A, CUBLAS_DATA_HALF, lda,
                  B, CUBLAS_DATA_HALF, ldb,
                  &beta,
                  C, CUBLAS_DATA_HALF, ldc);
#endif
}

void Nd4jBlas::sgemm(Nd4jPointer *extraParams, int Order, int TransA, int TransB,
                     int M, int N, int K,
                     float alpha,
                     float *A, int lda,
                     float *B, int ldb,
                     float beta,
                     float *C, int ldc) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSgemm(*handle, convertTranspose(TransA), convertTranspose(TransB), M, N, K, &alpha, A, lda,
                   B, ldb, &beta, C, ldc);

}

void Nd4jBlas::dgemm(Nd4jPointer *extraParams, int Order, int TransA, int TransB,
                     int M, int N, int K,
                     double alpha,
                     double *A, int lda,
                     double *B, int ldb,
                     double beta,
                     double *C, int ldc) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDgemm(*handle, convertTranspose(TransA), convertTranspose(TransB), M, N, K, &alpha, A, lda,
                   B, ldb, &beta, C, ldc);

}

/*
 * ------------------------------------------------------
 * SYMM
 * ------------------------------------------------------
 */

void Nd4jBlas::ssymm(Nd4jPointer *extraParams, int Order, int Side, int Uplo,
                     int M, int N,
                     float alpha,
                     float *A, int lda,
                     float *B, int ldb,
                     float beta,
                     float *C, int ldc) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsymm(*handle, convertSideMode(Side), convertUplo(Uplo), M, N, &alpha, A, lda, B, ldb, &beta,
                   C, ldc);

}

void Nd4jBlas::dsymm(Nd4jPointer *extraParams, int Order, int Side, int Uplo,
                     int M, int N,
                     double alpha,
                     double *A, int lda,
                     double *B, int ldb,
                     double beta,
                     double *C, int ldc) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsymm(*handle, convertSideMode(Side), convertUplo(Uplo), M, N, &alpha, A, lda, B, ldb, &beta,
                   C, ldc);

}

/*
 * ------------------------------------------------------
 * SYRK
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyrk(Nd4jPointer *extraParams, int Order, int Uplo, int Trans,
                     int N, int K,
                     float alpha,
                     float *A, int lda,
                     float beta,
                     float *C, int ldc) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsyrk(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, A, lda, &beta, C,
                   ldc);
}

void Nd4jBlas::dsyrk(Nd4jPointer *extraParams, int Order, int Uplo, int Trans,
                     int N, int K,
                     double alpha,
                     double *A, int lda,
                     double beta,
                     double *C, int ldc) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsyrk(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, A, lda, &beta, C,
                   ldc);

}

/*
 * ------------------------------------------------------
 * SYR2K
 * ------------------------------------------------------
 */

void Nd4jBlas::ssyr2k(Nd4jPointer *extraParams, int Order, int Uplo, int Trans,
                      int N, int K,
                      float alpha,
                      float *A, int lda,
                      float *B, int ldb,
                      float beta,
                      float *C, int ldc) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasSsyr2k(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, A, lda, B, ldb,
                    &beta, C, ldc);

}

void Nd4jBlas::dsyr2k(Nd4jPointer *extraParams, int Order, int Uplo, int Trans,
                      int N, int K,
                      double alpha,
                      double *A, int lda,
                      double *B, int ldb,
                      double beta,
                      double *C, int ldc) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDsyr2k(*handle, convertUplo(Uplo), convertTranspose(Trans), N, K, &alpha, A, lda, B, ldb,
                    &beta, C, ldc);

}

/*
 * ------------------------------------------------------
 * TRMM
 * ------------------------------------------------------
 */

void Nd4jBlas::strmm(Nd4jPointer *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     float alpha,
                     float *A, int lda,
                     float *B, int ldb) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    //float *C = reinterpret_cast<float *>(&extraParams[1]);
    hipblasStrmm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, A, lda, B, ldb, B, ldb);

}

void Nd4jBlas::dtrmm(Nd4jPointer *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     double alpha,
                     double *A, int lda,
                     double *B, int ldb) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtrmm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, A, lda, B, ldb, B, ldb);


}

/*
 * ------------------------------------------------------
 * TRSM
 * ------------------------------------------------------
 */

void Nd4jBlas::strsm(Nd4jPointer *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     float alpha,
                     float *A, int lda,
                     float *B, int ldb) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasStrsm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, A, lda, B, ldb);

}

void Nd4jBlas::dtrsm(Nd4jPointer *extraParams, int Order, int Side,
                     int Uplo, int TransA, int Diag,
                     int M, int N,
                     double alpha,
                     double *A, int lda,
                     double *B, int ldb) {
    hipblasHandle_t *handle = reinterpret_cast<hipblasHandle_t *>(&extraParams[0]);
    hipblasDtrsm(*handle, convertSideMode(Side), convertUplo(Uplo), convertTranspose(TransA), convertDiag(Diag), M, N,
                   &alpha, A, lda, B, ldb);


}

