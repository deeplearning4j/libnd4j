#include "hip/hip_runtime.h"
#include "../NativeOps.h"
#include <hip/hip_runtime.h>
#include <cuda_launch_config.h>

#include <buffer.h>
#include <shape.h>

#include <hipblas.h>
#include <reduce3.h>
#include <reduce.h>
#include <indexreduce.h>
#include <pairwise_transform.h>
#include <transform.h>
#include <scalar.h>
#include <broadcasting.h>
#include <summarystatsreduce.h>
#include <thread>
#include <map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <pointercast.h>
#include <stdio.h>

hipDeviceProp_t *deviceProperties;
hipFuncAttributes *funcAttributes = new hipFuncAttributes[28];
int blockLimit = 128;
int maxThreads = 512;
bool debug = false;
bool verbose = false;

template <typename T>
dim3 getOptimalDimensions(Nd4jIndex n,hipFuncAttributes attributes, hipDeviceProp_t properties) {

	// we can combine the two to compute a block size
	int num_threads = block_size_with_maximum_potential_occupancy(attributes, properties);

	// no real sense launching more threads, then number of elements we have
	if (num_threads > n) num_threads = n;

	if (maxThreads > 0 && num_threads > maxThreads) num_threads = maxThreads;

	// compute the number of blocks of size num_threads to launch
	int num_blocks = n / num_threads;

	// check for partial block at the end

	if (num_blocks > blockLimit) num_blocks = blockLimit;

	if (num_blocks < 4 && n > 128) {
		num_blocks = 4;
		num_threads = n / num_blocks;
	}

	if (num_threads >= 768) {
		num_blocks = num_blocks * 2;
		num_threads = num_threads / 2;
	}

	if(n % num_threads && num_blocks < blockLimit) ++num_blocks;
    //(num_threads * sizeof(T)) + attributes.sharedSizeBytes);
	return dim3(num_blocks,num_threads, 3000);
}

int getBaseMemorySize(int xRank, int yRank, int zRank) {
	int memory_limit = 1024;

	if (xRank == 0) xRank = 2;

	memory_limit += (xRank * 2 + 4) * 3; // we reserve memory for xShape + T1/T2 shapes
	memory_limit += yRank == 0 ? 0 : (yRank * 2 + 4);
	memory_limit += zRank == 0 ? 0 : (zRank * 2 + 4);
	memory_limit += xRank * 6;
	memory_limit += MAX_RANK; // special case, needed roughtly in one pase

	return memory_limit;
}

int getDeviceBlockThreshold(int deviceId) {
	int ccMinor = deviceProperties[deviceId].minor;
	int ccMajor = deviceProperties[deviceId].major;

	int blockThreshold;

	if (ccMajor >= 5)
		blockThreshold = 32;
	else if (ccMajor == 3)
		blockThreshold = 16;
	else if (ccMajor < 3)
		blockThreshold = 8;

	return blockThreshold;
}

dim3 getBasicLaunchParams(int deviceId, long problemLength, int sharedMemoryPerThread) {
	int countMP = deviceProperties[deviceId].multiProcessorCount;
	int blockThreshold = getDeviceBlockThreshold(deviceId);

	int num_threads = problemLength / (countMP * blockThreshold);
	num_threads = 64;

	int num_blocks = 64;

	int memory_limit = (sharedMemoryPerThread * num_threads) + getBaseMemorySize(1,0,0);

	dim3 launchDims = dim3(num_blocks, num_threads, memory_limit);

	if (debug && verbose)
		printf("Preliminary basic launch params: gridSize: [%i], blockSize: [%i], base shmem: [%i]\n", num_blocks, num_threads, memory_limit);


	return launchDims;
}

int getDeviceSharedThreshold(int deviceId) {
	int ccMinor = deviceProperties[deviceId].minor;
	int ccMajor = deviceProperties[deviceId].major;

	// please note threshold isn't multiple of 32, and that's NOT a mistake

	int shmemThreshold;
	if (ccMajor == 5 && ccMinor == 2)
		shmemThreshold = 96000;
	else if (ccMajor == 5)
		shmemThreshold = 64000;
	else if (ccMajor == 3 && ccMinor == 7)
		shmemThreshold = 112000;
	else shmemThreshold = 48000;

	return shmemThreshold;
}


dim3 getBetterDimensions(int deviceId, int numTads, int tadLength, int xRank, int yRank, int zRank, int dimensionLength, int elementSize, int reduction) {

	int num_threads = nd4j::math::nd4j_min<int>(tadLength, maxThreads);



	int countMP = deviceProperties[deviceId].multiProcessorCount;
	int regPerBlock = deviceProperties[deviceId].regsPerBlock;
	int warpSize = deviceProperties[deviceId].warpSize;

	int blockThreshold = getDeviceBlockThreshold(deviceId);
	int shmemThreshold = getDeviceSharedThreshold(deviceId);

	// round num_threads to nearest warpSize
	num_threads -= num_threads % warpSize;

	num_threads = nd4j::math::nd4j_max<int>(32, num_threads);


	// since we use shared memory as fast memory for some cases - we need to count that in
	int memory_limit = getBaseMemorySize(xRank, yRank, zRank);
	int memory_floor = memory_limit;
	int effective_block_limit =  countMP * blockThreshold;

	// at this moment we've stored all required information for things. time to count in reduction multipliers
	int reduction_per_block = 0;
	bool found = false;
	if (reduction > 0)
		while (!found) {
			reduction_per_block = (num_threads * elementSize * reduction);
			if (memory_limit + reduction_per_block < 5000) {
				memory_limit += reduction_per_block;
				found = true;
			} else {
				if (num_threads >= 128) {
					num_threads -= 32;
				} else {
					memory_limit += reduction_per_block;
					found = true;
				}
			}
		}

	// at this moment we know total memory used per block, and we also know per-mp limit.
	int max_active_blocks = shmemThreshold / memory_limit;

	// we don't want to spawn more blocks, that gpu can actually handle without queue
	int num_blocks = nd4j::math::nd4j_min<int>(numTads, effective_block_limit);
	num_blocks = nd4j::math::nd4j_min<int>(num_blocks, max_active_blocks);
	num_blocks = nd4j::math::nd4j_min<int>(num_blocks, blockLimit);
    num_blocks = num_blocks - (num_blocks % countMP);
	num_blocks = nd4j::math::nd4j_max<int>(num_blocks, 1);

	int targetBlocksPerMP = num_blocks / countMP;

	// now we know desired number of blocks wrt to shared memory. So, now we should take in account number of threads per SM
	if (targetBlocksPerMP * num_threads > 2048) {
		while (targetBlocksPerMP * num_threads > 2048) {
			if (num_threads <= 96)
				break;

			num_threads -= 32;
		}

		memory_limit = memory_floor + (num_threads * elementSize * reduction);
	}




	if (debug && verbose)
		printf("Preliminary reduce launch params: gridSize: [%i], blockSize: [%i], base shmem: [%i], reduction_per_block: [%i], blocksPerMP: [%i]\n", num_blocks, num_threads, memory_limit, reduction_per_block, targetBlocksPerMP);

	return dim3(num_blocks,num_threads, memory_limit);
}


dim3 getFlatLaunchParams(int deviceId, int *xShapeInfo, int *yShapeInfo) {
	int xRank = shape::rank(xShapeInfo);
	int yRank = yShapeInfo == nullptr ? 0 : shape::rank(yShapeInfo);
	int zRank = 0;

	int memory_limit = getBaseMemorySize(xRank, yRank, zRank);

	int countMP = deviceProperties[deviceId].multiProcessorCount;
	int regPerBlock = deviceProperties[deviceId].regsPerBlock;

	int blockThreshold = getDeviceBlockThreshold(deviceId);
	int shmemThreshold = getDeviceSharedThreshold(deviceId);

	int xLength = shape::length(xShapeInfo);


	int num_threads = nd4j::math::nd4j_min<int>(maxThreads, xLength);

	int memory_floor = memory_limit;
	int effective_block_limit =  countMP * blockThreshold;

	int num_blocks = xLength / num_threads;
	num_blocks = nd4j::math::nd4j_min<int>(num_blocks, blockLimit);
	num_blocks = nd4j::math::nd4j_min<int>(num_blocks, effective_block_limit);
	num_blocks = nd4j::math::nd4j_max<int>(num_blocks, 1);


	int targetBlocksPerMP = num_blocks / countMP;

	// now we know desired number of blocks wrt to shared memory. So, now we should take in account number of threads per SM
	if (targetBlocksPerMP * num_threads > 2048) {
		while (targetBlocksPerMP * num_threads > 2048) {
			num_threads -= 32;
			if (num_threads <= 96)
				break;
		}
	}

	dim3 launchDims = dim3(num_blocks, num_threads, memory_limit);

	if (debug && verbose)
		printf("Preliminary scalar launch params: gridSize: [%i], blockSize: [%i], base shmem: [%i], blocksPerMP: [%i], problemLength: [%i]\n", num_blocks, num_threads, memory_limit, targetBlocksPerMP, xLength);


	return launchDims;
}

dim3 getReduceLaunchParams(int deviceId, int *xShapeInfo, int *yShapeInfo, int *zShapeInfo, int dimensionLength, int elementSize, int reductionSize) {

	int tadLength = 0;
	int numTads = 0;
	if (zShapeInfo != nullptr) {
		tadLength = shape::length(xShapeInfo) / shape::length(zShapeInfo);
		numTads = shape::length(xShapeInfo) / tadLength;

		if (tadLength == 1) {
			if (debug && verbose)
				printf("xLength: [%i], zLength: [%i]\n", shape::length(xShapeInfo), shape::length(zShapeInfo));
		}
	} else{
		// we have special case - reduction along all dimensions
		if (debug && verbose)
			printf("zShapeInfo is nullPtr\n");
		tadLength = 2048;
		numTads = shape::length(xShapeInfo) / tadLength;
	}

	if (debug && verbose)
		printf("xLength: [%i], numTads: [%i], tadLength: [%i]\n", shape::length(xShapeInfo), numTads, tadLength);

	int xRank = shape::rank(xShapeInfo);
	int yRank = yShapeInfo == nullptr ? 0 : shape::rank(yShapeInfo);
	int zRank = zShapeInfo == nullptr ? 0 : shape::rank(zShapeInfo);

	return getBetterDimensions(deviceId, numTads, tadLength, xRank, yRank, zRank, dimensionLength, elementSize, reductionSize);
}

/**
 * Returns optimal launch parameters
 * given the extra pointers passed in.
 * The extra pointer should be
 * the host pointer for the shape information
 * associated with the data.
 * From there it is used to obtain the length
 * from which we can derive the optimal launch parameters.
 *
 */
template <typename T>
dim3 getOptimalLaunchParameters(Nd4jPointer *extraPointers, hipFuncAttributes attributes, hipDeviceProp_t properties) {
	int *hostXShapeInfo = reinterpret_cast<int *>(extraPointers[0]);
	Nd4jIndex n = shape::length(hostXShapeInfo);

	dim3 launchDims = getOptimalDimensions<T>(n,attributes, properties);

	if (debug && verbose)
		printf("Params: gridSize: [%i], blockSize: [%i], shMem: [%i], problemLength: [%i], totalThreads:[%i]\n", launchDims.x, launchDims.y, launchDims.z, n, (launchDims.x * launchDims.y));

	return launchDims;
}


nd4j::buffer::Buffer<int> * createScalarBuffer(hipStream_t stream) {
	int *scalarShapeInfo = shape::createScalarShapeInfo();
	nd4j::buffer::Buffer<int> *buff = nd4j::buffer::createBuffer(scalarShapeInfo,shape::shapeInfoLength(2), stream);
	nd4j::buffer::copyDataToGpu(&buff, stream);
	return buff;
}


class ScalarShapeInformation {
private:
	nd4j::buffer::Buffer<int> *scalarDimension;
	nd4j::buffer::Buffer<int> *scalarShapeInfo;
	std::thread::id threadId;

public:
	ScalarShapeInformation(hipStream_t stream) {
		int *scalarDimensionBuff = (int *) malloc(sizeof(int));
		scalarDimensionBuff[0] = MAX_DIMENSION;
		scalarDimension = nd4j::buffer::createBuffer(scalarDimensionBuff,1, stream);
		scalarShapeInfo = createScalarBuffer(stream);
		threadId = std::this_thread::get_id();

	}
	~ScalarShapeInformation() {
		nd4j::buffer::freeBuffer(&scalarShapeInfo);
		nd4j::buffer::freeBuffer(&scalarDimension);
	}


	int *getShapeInfoHostPointer() {
		return scalarShapeInfo->data;
	}

	int * getShapeInfoGpuPointer() {
		return scalarShapeInfo->gData;
	}

	int * getDimensionHostPointer() {
		return scalarDimension->data;
	}

	int  * getDimensionGpuPointer() {
		return scalarDimension->gData;
	}

};





template <typename T>
class ScalarInfo {
	nd4j::buffer::Buffer<T> *scalarData;
	ScalarShapeInformation *shapeInfo;
	T finalResult;
	hipStream_t streamRef;
public:
	ScalarInfo(hipStream_t stream) {
		T *scalarResult = (T*)malloc(sizeof(T));
		shapeInfo = new ScalarShapeInformation(stream);
		scalarData = nd4j::buffer::createBuffer(scalarResult,1, stream);
		streamRef = stream;
		nd4j::buffer::copyDataToGpu(&scalarData, stream);
	}

	T getFinalResultFromDevice() {
		nd4j::buffer::copyDataFromGpu(&scalarData, streamRef);
		return scalarData->data[0];
	}

	/**
	 * Get the device shape information
	 * representing a scalar
	 */
	 int *getDeviceShapeInfo() {
		return shapeInfo->getShapeInfoGpuPointer();
	}

	/**
	 * Get the result pointers
	 */
	 T *getDevicePointer() {
		 return scalarData->gData;
	 }

	 /**
	  * Get the infinite dimension device pointer
	  */
	 int *getDimensionDevicePointer() {
		 return shapeInfo->getDimensionGpuPointer();
	 }

	 ~ScalarInfo() {
		 nd4j::buffer::freeBuffer(&scalarData);
		 delete shapeInfo;
	 }
};


/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
double   NativeOps::execIndexReduceScalarDouble(Nd4jPointer *extraPointers,int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D1 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[27], deviceProperties[(int) extraPointers[2]]);

	double *resultPointer = reinterpret_cast<double *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, nullptr, 1, sizeof(double), 2);

	indexReduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			nullptr, 0,
			nullptr,
			1,
			1, allocationPointer, reductionPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	double result = resultPointer[0];
	return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduceDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension, int dimensionLength) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D2 opNum:[%i]\n", opNum);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, resultShapeInfoPointer, dimensionLength, sizeof(double), 2);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	indexReduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			dimensionPointer,
			dimensionLength,
			1, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcastDouble(Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer dimension, int dimensionLength){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D3 opNum:[%i]\n", opNum);

//	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[26], deviceProperties[(int) extraPointers[2]]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], resultShapeInfoPointer, yShapeInfoPointer,  dimensionLength, sizeof(double), 0);

	broadcastDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			yPointer,
			yShapeInfoPointer, shape::rank(yShapeInfoPointer),
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			dimensionPointer,
			dimensionLength);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}



/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param y
 * @param yStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execPairwiseTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		int xStride,
		Nd4jPointer y,
		int yStride,
		Nd4jPointer result,
		int resultStride,
		Nd4jPointer extraParams, Nd4jIndex n) {
	double *xPointer = reinterpret_cast<double *>(dx);
	double *yPointer = reinterpret_cast<double *>(y);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D4 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[25], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr);

	pairWiseTransformStridedDouble<<<launchDims.x,launchDims.y, launchDims.z, *stream>>> (
			opNum,
			n,
			xPointer,
			yPointer,
			xStride,
			yStride,
			extraParamsPointer,
			resultPointer,
			resultStride, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param yIndexes
 * @param resultIndexes
 */
void NativeOps::execPairwiseTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer xIndexes,
		Nd4jPointer yIndexes,
		Nd4jPointer resultIndexes) {
	double *xPointer = reinterpret_cast<double *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
	int *yIndexesPointer = reinterpret_cast<int *>(yIndexes);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D5 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[24], deviceProperties[(int) extraPointers[2]]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], resultShapeInfoPointer);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	pairWiseTransformDoubleIndex <<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(
			opNum,
			xPointer,
			yPointer,
			extraParamsPointer,
			resultPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			yShapeInfoPointer, shape::rank(yShapeInfoPointer),
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			xIndexesPointer,
			yIndexesPointer,
			resultIndexesPointer, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void NativeOps::execPairwiseTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer  xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer  yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer  resultShapeInfo,
		Nd4jPointer extraParams) {
	double *xPointer = reinterpret_cast<double *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D6 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[23], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], resultShapeInfoPointer);

	pairWiseTransformDouble<<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(
			opNum,
			xPointer,
			yPointer,
			extraParamsPointer,
			resultPointer,
			xShapeInfoPointer,  shape::rank(xShapeInfoPointer),
			yShapeInfoPointer,  shape::rank(yShapeInfoPointer),
			resultShapeInfoPointer,  shape::rank(resultShapeInfoPointer), allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D7 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[22], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, resultShapeInfoPointer, 1, sizeof(double), 1);

	reduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			nullptr,
			1,
			1,
			allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceDouble(
		Nd4jPointer *extraPointers
		,int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer dimension,
		int dimensionLength) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D8 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[22], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, resultShapeInfoPointer, dimensionLength, sizeof(double), 1);

	reduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			dimensionPointer,
			dimensionLength,
			1,
			allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @return
 */
double NativeOps::execReduceScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D9 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[22], deviceProperties[(int) extraPointers[2]]);

	double *resultPointer = reinterpret_cast<double *>(extraPointers[5]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, nullptr, 1, sizeof(double), 1);

	reduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			nullptr, 0,
			nullptr,
			1,
			1,
			allocPointer, reductionPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	double result = resultPointer[0];
	return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduce3Double(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D10 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[21], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	//dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer, resultShapeInfoPointer, 1, sizeof(double), 2);
	//dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer);
	dim3 launchDims = getBasicLaunchParams((int) extraPointers[2], shape::length((int *) extraPointers[0]), 16);

	reduce3Double<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			nullptr,
			1,
			1, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 */
double   NativeOps::execReduce3ScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[21], deviceProperties[(int) extraPointers[2]]);

	if (debug && verbose)
		printf("D11 opNum:[%i]\n", opNum);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	double *resultPointer = reinterpret_cast<double *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	//dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer, nullptr, 1, sizeof(double), 2);
	//dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer);
	dim3 launchDims = getBasicLaunchParams((int) extraPointers[2], shape::length((int *) extraPointers[0]), 16);

	reduce3Double<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			nullptr,
			nullptr,
			1,
			1, allocationPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	double result  = resultPointer[0];
	return result;
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execReduce3Double(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension,
		int dimensionLength){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *yPointer = reinterpret_cast<double *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParamsVals);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D12 opNum:[%i]\n", opNum);

//	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[21], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	//dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer, resultShapeInfoPointer, dimensionLength, sizeof(double), 2);
	//dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer);
	dim3 launchDims = getBasicLaunchParams((int) extraPointers[2], shape::length((int *) extraPointers[0]), 16);

	reduce3Double<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength,
			1, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}
/**
 *
 * @param opNum
 * @param x
 * @param xStride
 * @param result
 * @param resultStride
 * @param scalar
 * @param extraParams
 * @param n
 */
void   NativeOps::execScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		int xStride,
		Nd4jPointer result,
		int resultStride,
		double scalar,
		Nd4jPointer extraParams,
		Nd4jIndex n) {
	double *xPointer = reinterpret_cast<double *>(x);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D13 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[20], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr);

	scalarDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			scalar,
			xPointer,
			xStride,
			extraParamsPointer,
			resultPointer,resultStride, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 */
void NativeOps::execScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		double scalar,
		Nd4jPointer extraParams){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D14 opNum:[%i]\n", opNum);

//	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[19], deviceProperties[(int) extraPointers[2]]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], resultShapeInfoPointer);

	scalarDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			scalar,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,resultShapeInfoPointer, shape::rank(resultShapeInfoPointer), allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param resultIndexes
 */
void NativeOps::execScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		double scalar,
		Nd4jPointer extraParams,
		Nd4jIndex n,
		Nd4jPointer xIndexes,
		Nd4jPointer resultIndexes){
	double *xPointer = reinterpret_cast<double *>(x);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D15 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[18], deviceProperties[(int) extraPointers[2]]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], (int *) resultShapeInfo);

	scalarDoubleIndexes<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			scalar,
			xPointer,
			extraParamsPointer,
			resultPointer,
			resultIndexesPointer, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
double   NativeOps::execSummaryStatsScalarDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,bool biasCorrected){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D16 opNum:[%i]\n", opNum);

//	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[17], deviceProperties[(int) extraPointers[2]]);

	double *resultPointer = reinterpret_cast<double *>(extraPointers[5]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, nullptr, 1, sizeof(double), 8);

	summaryStatsReduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			nullptr, 0,
			nullptr,
			1,
			1,biasCorrected, allocationPointer, reductionPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	double result = resultPointer[0];
	return result;

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execSummaryStatsDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,bool biasCorrected) {
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	if (debug && verbose)
		printf("D17 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[17], deviceProperties[(int) extraPointers[2]]);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, resultShapeInfoPointer, 1, sizeof(double), 8);

	summaryStatsReduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			nullptr,
			1,
			1,biasCorrected, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execSummaryStatsDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension, int dimensionLength,bool biasCorrected){
	double *xPointer = reinterpret_cast<double *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D18 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[17], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, resultShapeInfoPointer, dimensionLength, sizeof(double), 8);

	summaryStatsReduceDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			dimensionPointer,
			dimensionLength,
			1,biasCorrected, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		int xStride,
		Nd4jPointer result,
		int resultStride,
		Nd4jPointer extraParams,
		Nd4jIndex n) {
	double *xPointer = reinterpret_cast<double *>(dx);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D19 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[16], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr);

	transformDouble<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			xPointer,
			xStride,
			extraParamsPointer,
			resultPointer,resultStride, allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams){
	double *xPointer = reinterpret_cast<double *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *resultShapeInfoPointer =  reinterpret_cast<int *>(resultShapeInfo);
	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	int *hostXShapeInfo = reinterpret_cast<int *>(extraPointers[0]);

	if (debug && verbose)
		printf("D20 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[1], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	// special pointer for special buffer for special ops
	double *specialPointer = reinterpret_cast<double *>(extraPointers[6]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], resultShapeInfoPointer);

	int *dimension = (int *) allocPointer;
	int *maxDimension = dimension + 1;
	int *maxShapeBuffer = (int *) maxDimension + 1;
	double * special = (double *) maxShapeBuffer + 8;

	// simple trick to get workaround over reductions into scalar
	if (opNum >= 38 && opNum <= 41) {
		if (shape::isVector(hostXShapeInfo) && opNum != 41) {
			// if that's vector, we just go directly to op in 1 block
			int length = shape::length(hostXShapeInfo);
			transformDouble<<< 1, nd4j::math::nd4j_min<int>(512, length), launchDims.z, *stream >>> (
					opNum,
							xPointer,
							xShapeInfoPointer,  shape::rank(xShapeInfoPointer),
							extraParamsPointer,
							resultPointer, resultShapeInfoPointer,  shape::rank(resultShapeInfoPointer), allocPointer, reductionPointer);
		} else {
			// going for blockwise specials
			//float *xpf = reinterpret_cast<float *>(dx);

			int *shape = shape::shapeOf(hostXShapeInfo);
			//printf("Rows num: %i\n", shape[0]);
			switch (opNum) {
				case 40: // LogSoftMax
				case 39: // SoftMax Derivative
				case 38: {// softmax
					prepareShapeBuffer << < 1, 1, 128, *stream >> > (dimension, maxDimension, maxShapeBuffer, shape[0]);

					//checkCudaErrors(hipStreamSynchronize(*stream));

					// max 3
					execReduceDouble(extraPointers, 3, dx, xShapeInfo, extraParams, (Nd4jPointer) special,
									(Nd4jPointer) maxShapeBuffer, (Nd4jPointer) maxDimension, 1);

					// sub 1
					execBroadcastDouble(extraPointers, 1, dx, xShapeInfo, (Nd4jPointer) special,
									   (Nd4jPointer) maxShapeBuffer, dx, xShapeInfo, (Nd4jPointer) dimension, 1);

					// exp 3
					execTransformDouble(extraPointers, 3, dx, xShapeInfo, dx, xShapeInfo, extraParams);

					//sum 1
					execReduceDouble(extraPointers, 1, dx, xShapeInfo, extraParams, (Nd4jPointer) special,
									(Nd4jPointer) maxShapeBuffer, (Nd4jPointer) maxDimension, 1);

					// divide 3
					execBroadcastDouble(extraPointers, 3, dx, xShapeInfo, (Nd4jPointer) special,
									   (Nd4jPointer) maxShapeBuffer, dx, xShapeInfo, (Nd4jPointer) dimension, 1);

					// log 3
					if (opNum == 40)
						execTransformDouble(extraPointers, 5, dx, xShapeInfo, dx, xShapeInfo, extraParams);
					else if (opNum == 39)
						execTransformDouble(extraPointers, 42, dx, xShapeInfo, dx, xShapeInfo, extraParams);

					break;
				}
				case 41: {
					// IsMax along all dimensions
					if (extraParamsPointer == nullptr) {
						int maxIdx = (int) execIndexReduceScalarDouble(extraPointers, 0, dx, xShapeInfo, extraParams);
						int targetIdx = 0;

						if (shape::order(hostXShapeInfo) == 'c' || shape::order(hostXShapeInfo) == 'f' && maxIdx * shape::stride(hostXShapeInfo)[shape::rank(hostXShapeInfo) - 1] >= shape::length(hostXShapeInfo))
							targetIdx = maxIdx;
						else
							targetIdx = maxIdx * shape::stride(hostXShapeInfo)[shape::rank(hostXShapeInfo) - 1];

						fillIsMaxDouble<<< 8, 96, 0, *stream >>>(resultPointer, shape::length(hostXShapeInfo), targetIdx);
					} else {
						// going for dimension-based IsMax
						execIndexReduceDouble(extraPointers,0, dx, xShapeInfo, extraParams, result, resultShapeInfo, (Nd4jPointer) dimension, 1);
					}
					break;
				}
				default: {
					printf("Bad case for transformFloat\n");
					break;
				}
			}
		}
	} else {
		transformDouble<<<launchDims.x, launchDims.y, launchDims.z, *stream>>> (
				opNum,
						xPointer,
						xShapeInfoPointer,  shape::rank(xShapeInfoPointer),
						extraParamsPointer,
						resultPointer, resultShapeInfoPointer, shape::rank(resultShapeInfoPointer), allocPointer, reductionPointer);
	}
	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformDouble(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer xIndexes,
		Nd4jPointer resultIndexes) {
	double *xPointer = reinterpret_cast<double *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	double *resultPointer = reinterpret_cast<double *>(result);
	double *extraParamsPointer = reinterpret_cast<double *>(extraParams);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D21 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[14], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	double *reductionPointer = reinterpret_cast<double *>(extraPointers[4]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], (int *) resultShapeInfo);

	transformDoubleIndexes<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultIndexesPointer, allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
float   NativeOps::execIndexReduceScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	if (debug && verbose)
		printf("F1 opNum:[%i]\n", opNum);

//	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[13], deviceProperties[(int) extraPointers[2]]);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	float *resultPointer = reinterpret_cast<float *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, nullptr, 1, sizeof(float), 2);

	indexReduceFloat<<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			nullptr, 0,
			nullptr,
			1,
			1, allocationPointer, reductionPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	float result = resultPointer[0];
	return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execIndexReduceFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension,
		int dimensionLength){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F2 opNum:[%i]\n", opNum);

	// dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[13], deviceProperties[(int) extraPointers[2]]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, resultShapeInfoPointer, dimensionLength, sizeof(float), 2);

	indexReduceFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			dimensionPointer,
			dimensionLength,
			1, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execBroadcastFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer dimension, int dimensionLength){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F3 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[12], deviceProperties[(int) extraPointers[2]]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], resultShapeInfoPointer, yShapeInfoPointer, 1, sizeof(float), 0);

	broadcastFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			yPointer,
			yShapeInfoPointer, shape::rank(yShapeInfoPointer),
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			dimensionPointer,
			dimensionLength);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}



/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param y
 * @param yStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execPairwiseTransformFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		int xStride,
		Nd4jPointer y,
		int yStride,
		Nd4jPointer result,
		int resultStride,
		Nd4jPointer extraParams, Nd4jIndex n){
	float *xPointer = reinterpret_cast<float *>(dx);
	float *yPointer = reinterpret_cast<float *>(y);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F4 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[11], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, nullptr, 1, sizeof(float), 0);

	pairWiseTransformStridedFloat<<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(
			opNum,
			n,
			xPointer,
			yPointer,
			xStride,
			yStride,
			extraParamsPointer,
			resultPointer,
			resultStride, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param yIndexes
 * @param resultIndexes
 */
void NativeOps::execPairwiseTransformFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer xIndexes,
		Nd4jPointer yIndexes,
		Nd4jPointer resultIndexes){
	float *xPointer = reinterpret_cast<float *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *xIndexesPointer = reinterpret_cast<int *>(xIndexes);
	int *yIndexesPointer = reinterpret_cast<int *>(yIndexes);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F5 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[10], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer, resultShapeInfoPointer, 1, sizeof(float), 0);

	pairWiseTransformFloatIndex<<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(
			opNum,
			xPointer,
			yPointer,
			extraParamsPointer,
			resultPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			yShapeInfoPointer, shape::rank(yShapeInfoPointer),
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			xIndexesPointer,
			yIndexesPointer,
			resultIndexesPointer, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void NativeOps::execPairwiseTransformFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer  xShapeInfo,
		Nd4jPointer y,
		Nd4jPointer  yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer  resultShapeInfo,
		Nd4jPointer extraParams){
	float *xPointer = reinterpret_cast<float *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F6 opNum:[%i]\n", opNum);

//	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[9], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	//dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], resultShapeInfoPointer,  yShapeInfoPointer, 1, sizeof(float), 0);
	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], resultShapeInfoPointer);

	pairWiseTransformFloat<<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(
			opNum,
			xPointer,
			yPointer,
			extraParamsPointer,
			resultPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			yShapeInfoPointer, shape::rank(yShapeInfoPointer),
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer), allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo) {
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F7 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[8], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, resultShapeInfoPointer, 1, sizeof(float), 1);

	reduceFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			nullptr,
			1,
			1,
			allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduceFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer dimension,int dimensionLength){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F8 opNum:[%i]\n", opNum);

//	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[8], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);


	// DO NOT REMOVE COMMENTS OR CODE BELOW.
	// raver119@gmail.com

//	shape::TAD *tad = new shape::TAD();
//	tad->init(xShapeInfoPointer, dimensionPointer, dimensionLength);
//	tad->setOutputBuffer(allocPointer);
//	tad->createTadOnlyShapeInfo();

//	shape::printShapeInfo(tad->tadOnlyShapeInfo);

// dim3 getBetterDimensions(int deviceId, int numTads, int tadLength, int xRank, int yRank, int zRank, int dimensionLength, int elementSize, int reduction)

	dim3 temp = getReduceLaunchParams((int) extraPointers[2], xShapeInfoPointer, nullptr, resultShapeInfoPointer, dimensionLength, sizeof(float), 1);

	reduceFloat<<<temp.x,temp.y,temp.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			dimensionPointer,
			dimensionLength,
			1,
			allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

	//delete tad;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @return
 */
float NativeOps::execReduceScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F9 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[8], deviceProperties[(int) extraPointers[2]]);

	float *resultPointer = reinterpret_cast<float *>(extraPointers[5]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	dim3 temp = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, nullptr, 1, sizeof(float), 1);

	reduceFloat<<< temp.x,temp.y, temp.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			nullptr, 0,
			nullptr,
			1,
			1,
			allocPointer,
			reductionPointer
	);


	checkCudaErrors(hipStreamSynchronize(*stream));

	float result = resultPointer[0];
	return result;
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execReduce3Float(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F10 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[7], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	//dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer, resultShapeInfoPointer, 1, sizeof(float), 2);
	//dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer);
	dim3 launchDims = getBasicLaunchParams((int) extraPointers[2], shape::length((int *) extraPointers[0]), 16);

	reduce3Float<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			nullptr,
			1,
			1, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 */
float   NativeOps::execReduce3ScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo) {
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F11 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[7], deviceProperties[(int) extraPointers[2]]);

	float *resultPointer = reinterpret_cast<float *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	//dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer, nullptr, 1, sizeof(float), 2);
	//dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer);
	dim3 launchDims = getBasicLaunchParams((int) extraPointers[2], shape::length((int *) extraPointers[0]), 16);

	reduce3Float<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			nullptr,
			nullptr,
			1,
			1, allocationPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	double result  = resultPointer[0];
	return result;

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParamsVals
 * @param y
 * @param yShapeInfo
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execReduce3Float(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParamsVals,
		Nd4jPointer y,
		Nd4jPointer yShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension,
		int dimensionLength){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *yPointer = reinterpret_cast<float *>(y);
	int *yShapeInfoPointer = reinterpret_cast<int *>(yShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParamsVals);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[7], deviceProperties[(int) extraPointers[2]]);

	if (debug && verbose)
		printf("F12 opNum:[%i]\n", opNum);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);

	//dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer, resultShapeInfoPointer, dimensionLength, sizeof(float), 2);
	//dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], yShapeInfoPointer);
	dim3 launchDims = getBasicLaunchParams((int) extraPointers[2], shape::length((int *) extraPointers[0]), 16);

	reduce3Float<<<1,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,
			yPointer,
			yShapeInfoPointer,
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer,
			dimensionPointer,
			dimensionLength,
			1, allocationPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param x
 * @param xStride
 * @param result
 * @param resultStride
 * @param scalar
 * @param extraParams
 * @param n
 */
void   NativeOps::execScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		int xStride,
		Nd4jPointer result,
		int resultStride,
		double scalar,
		Nd4jPointer extraParams,
		Nd4jIndex n){
	float *xPointer = reinterpret_cast<float *>(x);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	if (debug && verbose)
		printf("F13 opNum:[%i]\n", opNum);

	dim3 temp = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr);

	scalarFloat<<<temp.x,temp.y,temp.z, *stream>>>(
			opNum,
			n,
			scalar,
			xPointer,
			xStride,
			extraParamsPointer,
			resultPointer,resultStride, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 */
void NativeOps::execScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		float scalar,
		Nd4jPointer extraParams){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	int *hostXShapeInfo = reinterpret_cast<int *>(extraPointers[0]);
	Nd4jIndex n = shape::length(hostXShapeInfo);

	if (debug && verbose)
		printf("F14 opNum:[%i]\n", opNum);

	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[5], deviceProperties[(int) extraPointers[2]]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 temp = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], resultShapeInfoPointer);

	scalarFloat<<<temp.x, temp.y,temp.z, *stream>>>(
			opNum,
			scalar,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,resultShapeInfoPointer, shape::rank(resultShapeInfoPointer), allocPointer );

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param scalar
 * @param extraParams
 * @param n
 * @param xIndexes
 * @param resultIndexes
 */
void NativeOps::execScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		double scalar,
		Nd4jPointer extraParams,
		Nd4jPointer xIndexes,
		Nd4jPointer resultIndexes){
	float *xPointer = reinterpret_cast<float *>(x);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);
	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	int *hostShapeInfo = reinterpret_cast<int *>(extraPointers[0]);
	Nd4jIndex n = shape::length(hostShapeInfo);

	if (debug && verbose)
		printf("F15 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[4], deviceProperties[(int) extraPointers[2]]);
	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], (int *) resultShapeInfo);

	scalarFloatIndexes<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			scalar,
			xPointer,
			extraParamsPointer,
			resultPointer,
			resultIndexesPointer, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 */
float   NativeOps::execSummaryStatsScalarFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,bool biasCorrected){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F16 opNum:[%i]\n", opNum);

//	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[3], deviceProperties[(int) extraPointers[2]]);

	float *resultPointer = reinterpret_cast<float *>(extraPointers[5]);
	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, nullptr, 1, sizeof(float), 8);

	summaryStatsReduceFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			nullptr, 0,
			nullptr,
			1,
			1,biasCorrected, allocationPointer, reductionPointer);

	checkCudaErrors(hipStreamSynchronize(*stream));

	float result = resultPointer[0];
	return result;
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfo
 */
void   NativeOps::execSummaryStatsFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,bool biasCorrected){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F17 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[3], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, resultShapeInfoPointer, 1, sizeof(float), 8);

	summaryStatsReduceFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			nullptr,
			1,
			1,biasCorrected, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 *
 * @param opNum
 * @param x
 * @param xShapeInfo
 * @param extraParams
 * @param result
 * @param resultShapeInfoBuffer
 * @param dimension
 * @param dimensionLength
 */
void   NativeOps::execSummaryStatsFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer x,
		Nd4jPointer xShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfoBuffer,
		Nd4jPointer dimension,
		int dimensionLength,bool biasCorrected){
	float *xPointer = reinterpret_cast<float *>(x);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfoBuffer);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *dimensionPointer = reinterpret_cast<int *>(dimension);
	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F18 opNum:[%i]\n", opNum);

	//	dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[3], deviceProperties[(int) extraPointers[2]]);

	int *allocationPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	dim3 launchDims = getReduceLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr, resultShapeInfoPointer, dimensionLength, sizeof(float), 8);

	summaryStatsReduceFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer, shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultShapeInfoPointer, shape::rank(resultShapeInfoPointer),
			dimensionPointer,
			dimensionLength,
			1,biasCorrected, allocationPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}
/**
 *
 * @param opNum
 * @param dx
 * @param xStride
 * @param result
 * @param resultStride
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		int xStride,
		Nd4jPointer result,
		int resultStride,
		Nd4jPointer extraParams,
		Nd4jIndex n) {
	float *xPointer = reinterpret_cast<float *>(dx);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F19 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[2], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr);

	transformFloat<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			n,
			xPointer,
			xStride,
			extraParamsPointer,
			resultPointer,resultStride, allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(Nd4jPointer *extraPointers,int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams) {
	float *xPointer = reinterpret_cast<float *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *resultShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);
	int *hostXShapeInfo = reinterpret_cast<int *>(extraPointers[0]);

	if (debug && verbose)
		printf("F20 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[1], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	// special pointer for special buffer for special ops
	float *specialPointer = reinterpret_cast<float *>(extraPointers[6]);

	int *dimension = (int *) allocPointer;
	int *maxDimension = dimension + 1;
	int *maxShapeBuffer = (int *) maxDimension + 1;
	float * special = (float *) maxShapeBuffer + 8;

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], resultShapeInfoPointer);

	// simple trick to get workaround over reductions into scalar
	if (opNum >= 38 && opNum <= 41) {
		if (shape::isVector(hostXShapeInfo) && opNum != 41) {
			// if that's vector, we just go directly to op in 1 block
			int length = shape::length(hostXShapeInfo);
			int block = nd4j::math::nd4j_min<int>(length, 256);
			transformFloat <<< 1, block, launchDims.z + (block * sizeof(float) * 4), *stream >> > (
					opNum,
					xPointer,
					xShapeInfoPointer,  shape::rank(xShapeInfoPointer),
					extraParamsPointer,
					resultPointer, resultShapeInfoPointer,  shape::rank(resultShapeInfoPointer),  allocPointer, reductionPointer);
		} else {
			// going for blockwise specials
			//float *xpf = reinterpret_cast<float *>(dx);

			int *shape = shape::shapeOf(hostXShapeInfo);
			switch (opNum) {
				case 40: // LogSoftMax
				case 39: // SoftMax Derivative
				case 38: {// softmax
					prepareShapeBuffer << < 1, 1, 128, *stream >> > (dimension, maxDimension, maxShapeBuffer, shape[0]);

					if (debug)
						checkCudaErrors(hipStreamSynchronize(*stream));

					//shape::printShapeInfo(maxShapeBuffer);

					// max 3
					execReduceFloat(extraPointers, 3, dx, xShapeInfo, extraParams, (Nd4jPointer) special,
									(Nd4jPointer) maxShapeBuffer, (Nd4jPointer) maxDimension, 1);

					// sub 1
					execBroadcastFloat(extraPointers, 1, dx, xShapeInfo, (Nd4jPointer) special,
									   (Nd4jPointer) maxShapeBuffer, dx, xShapeInfo, (Nd4jPointer) dimension, 1);

					// exp 3
					execTransformFloat(extraPointers, 3, dx, xShapeInfo, dx, xShapeInfo, extraParams);

					//sum 1
					execReduceFloat(extraPointers, 1, dx, xShapeInfo, extraParams, (Nd4jPointer) special,
									(Nd4jPointer) maxShapeBuffer, (Nd4jPointer) maxDimension, 1);

					// divide 3
					execBroadcastFloat(extraPointers, 3, dx, xShapeInfo, (Nd4jPointer) special,
									   (Nd4jPointer) maxShapeBuffer, dx, xShapeInfo, (Nd4jPointer) dimension, 1);

					// log 3
					if (opNum == 40)
						execTransformFloat(extraPointers, 5, dx, xShapeInfo, dx, xShapeInfo, extraParams);
					else if (opNum == 39)
						execTransformFloat(extraPointers, 42, dx, xShapeInfo, dx, xShapeInfo, extraParams);

					break;
				}
				case 41: {
					// IsMax along all dimensions
					if (extraParamsPointer == nullptr) {
						int maxIdx = (int) execIndexReduceScalarFloat(extraPointers, 0, dx, xShapeInfo, extraParams);
						int targetIdx = 0;

						if (shape::order(hostXShapeInfo) == 'c' || shape::order(hostXShapeInfo) == 'f' && maxIdx * shape::stride(hostXShapeInfo)[shape::rank(hostXShapeInfo) - 1] >= shape::length(hostXShapeInfo))
							targetIdx = maxIdx;
						else
							targetIdx = maxIdx * shape::stride(hostXShapeInfo)[shape::rank(hostXShapeInfo) - 1];

						fillIsMaxFloat<<< 256, 256, 0, *stream >>>(resultPointer, shape::length(hostXShapeInfo), targetIdx);
					} else {
						// going for dimension-based IsMax
					}
					break;
				}
				default: {
					printf("Bad case for transformFloat\n");
					break;
				}
			}
		}
	} else {
		transformFloat <<<launchDims.x, launchDims.y, launchDims.z, *stream>>> (
				opNum,
				xPointer,
				xShapeInfoPointer,  shape::rank(xShapeInfoPointer),
				extraParamsPointer,
				resultPointer, resultShapeInfoPointer,  shape::rank(resultShapeInfoPointer), allocPointer, reductionPointer);
	}

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));

}

/**
 *
 * @param opNum
 * @param dx
 * @param xShapeInfo
 * @param result
 * @param resultShapeInfo
 * @param extraParams
 * @param n
 */
void   NativeOps::execTransformFloat(
		Nd4jPointer *extraPointers,
		int opNum,
		Nd4jPointer dx,
		Nd4jPointer xShapeInfo,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer extraParams,
		Nd4jPointer xIndexes,
		Nd4jPointer resultIndexes) {
	float *xPointer = reinterpret_cast<float *>(dx);
	int *xShapeInfoPointer = reinterpret_cast<int *>(xShapeInfo);
	float *resultPointer = reinterpret_cast<float *>(result);
	float *extraParamsPointer = reinterpret_cast<float *>(extraParams);
	int *resultIndexesPointer = reinterpret_cast<int *>(resultIndexes);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("F21 opNum:[%i]\n", opNum);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[0], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);
	float *reductionPointer = reinterpret_cast<float *>(extraPointers[4]);

	dim3 launchDims = getFlatLaunchParams((int) extraPointers[2], (int *) extraPointers[0], nullptr);

	transformFloatIndexes<<<launchDims.x,launchDims.y,launchDims.z, *stream>>>(
			opNum,
			xPointer,
			xShapeInfoPointer,  shape::rank(xShapeInfoPointer),
			extraParamsPointer,
			resultPointer,
			resultIndexesPointer, allocPointer, reductionPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));


}


template <typename T>
__device__ void flattenKernelGeneric(int dOffset,
					char order,
					T *result,
					int *resultShapeInfo,
					T *input,
					int *inputShapeInfo, int *allocationPointer) {

	__shared__ UnifiedSharedMemory<T> *manager;

	if (threadIdx.x == 0) {
		extern __shared__ unsigned char shmem[];
		manager = new(shmem) UnifiedSharedMemory<T>();
		manager->init(sizeof(UnifiedSharedMemory<T>), 4, 4, sizeof(shape::TAD));
	}
	__syncthreads();

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int *zShape = shape::shapeOf(resultShapeInfo);
	int *zStride = shape::stride(resultShapeInfo);


	int *yShape = shape::shapeOf(inputShapeInfo);
	int *yStride = shape::stride(inputShapeInfo);
	char yOrder = shape::order(inputShapeInfo);

	int len = shape::length(inputShapeInfo);

	int resultEWS = shape::elementWiseStride(resultShapeInfo);
	int inputEWS = shape::elementWiseStride(inputShapeInfo);
	if (yOrder == order) {
		if (resultEWS >= 1 && inputEWS >= 1) {
			for (int i = tid; i < len; i+= gridDim.x * blockDim.x) {
				result[i * resultEWS + dOffset] = input[i * inputEWS];
			}
		} else {

			int rank = shape::rank(inputShapeInfo);
			/*
			long allocSize = sizeof(int) * rank;
			int *coord = shape::cuMalloc(allocationPointer, allocSize, manager);
			 */
			int coord[MAX_RANK];

			if(order == 'f') {
				for(int i = tid; i < len; i+= gridDim.x * blockDim.x) {
					shape::ind2sub(rank,yShape,i,coord);
					int offset = shape::getOffset(0,yShape,yStride,coord,rank);
					result[i + dOffset] = input[offset];
				}
			}
			else {
				for(int i = tid; i < len; i+= gridDim.x * blockDim.x) {
					shape::ind2subC(rank,yShape,i,coord);
					int offset = shape::getOffset(0,yShape,yStride,coord,rank);
					result[i + dOffset] = input[offset];
				}
			}
/*
			if (rank > MAX_COORD && tid * allocSize > PREALLOC_SIZE - allocSize) {
				free(coord);
			}
			*/
		}
	} else {
		int rank = shape::rank(inputShapeInfo);
		/*
		long allocSize = sizeof(int) * rank;
		int *coord = shape::cuMalloc(allocationPointer, allocSize, manager);
		 */
		int coord[MAX_RANK];

		if(order == 'f') {
			for(int i = tid; i < len; i+= gridDim.x * blockDim.x) {
				shape::ind2sub(rank,yShape,i,coord);
				int offset = shape::getOffset(0,yShape,yStride,coord,rank);
				result[i+dOffset] = input[offset];
			}
		}
		else {
			for(int i = tid; i < len; i+= gridDim.x * blockDim.x) {
				shape::ind2subC(rank,yShape,i,coord);
				int offset = shape::getOffset(0,yShape,yStride,coord,rank);
				result[i+dOffset] = input[offset];
			}
		}
		/*
		if (rank > MAX_COORD && tid * allocSize > PREALLOC_SIZE - allocSize) {
			free(coord);
		}*/
	}

}

extern "C" __global__ void flattenKernelDouble(int offset,
											  char order,
											  double *result,
											  int *resultShapeInfo,
											  double *input,
											  int *inputShapeInfo, int *allocationPointer) {
	flattenKernelGeneric<double>(offset, order, result, resultShapeInfo, input, inputShapeInfo, allocationPointer);
}

extern "C" __global__ void flattenKernelFloat(int offset,
											  char order,
											  float *result,
											  int *resultShapeInfo,
											  float *input,
											  int *inputShapeInfo, int *allocationPointer) {

	flattenKernelGeneric<float>(offset, order, result, resultShapeInfo, input, inputShapeInfo, allocationPointer);
}

/**
 * Append an input array
 * to the end of a flat array
 * in a particular order
 * @param offset the offset of the array to start at
 * @param order the order
 * @param result the result array
 * @param resultShapeInfo the shape info for te array
 * @param input the input for the array
 * @param inputShapeInfo the shape information for that array
 */
void NativeOps::flattenFloat(
		Nd4jPointer *extraPointers,
		int offset,
		char order,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer input,
		Nd4jPointer inputShapeInfo) {
	float *xPointer = reinterpret_cast<float *>(result);
	int *xShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	float *yPointer = reinterpret_cast<float *>(input);
	int *yShapeInfoPointer = reinterpret_cast<int *>(inputShapeInfo);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[5], deviceProperties[(int) extraPointers[2]]);

	if (debug && verbose)
		printf("F22 opNum:[7]\n");

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getBasicLaunchParams((int) extraPointers[2], shape::length(yShapeInfoPointer), 2);

	flattenKernelFloat<<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(offset, order, xPointer, xShapeInfoPointer, yPointer, yShapeInfoPointer, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}
/**
 * Append an input array
 * to the end of a flat array
 * in a particular order
 * @param offset the offset of the array to start at
 * @param order the order
 * @param result the result array
 * @param resultShapeInfo the shape info for te array
 * @param input the input for the array
 * @param inputShapeInfo the shape information for that array
 */
void NativeOps::flattenDouble(
		Nd4jPointer *extraPointers,
		int offset,
		char order,
		Nd4jPointer result,
		Nd4jPointer resultShapeInfo,
		Nd4jPointer input,
		Nd4jPointer inputShapeInfo) {
	double *xPointer = reinterpret_cast<double *>(result);
	int *xShapeInfoPointer = reinterpret_cast<int *>(resultShapeInfo);
	double *yPointer = reinterpret_cast<double *>(input);
	int *yShapeInfoPointer = reinterpret_cast<int *>(inputShapeInfo);

	hipStream_t *stream = reinterpret_cast<hipStream_t *>(&extraPointers[1]);

	if (debug && verbose)
		printf("D30 opNum:[7]\n");

	//dim3 launchDims = getOptimalLaunchParameters<float>(&extraPointers[0], funcAttributes[5], deviceProperties[(int) extraPointers[2]]);

	int *allocPointer = reinterpret_cast<int *>(extraPointers[3]);

	dim3 launchDims = getBasicLaunchParams((int) extraPointers[2], shape::length(yShapeInfoPointer), 2);

	flattenKernelDouble<<<launchDims.x,launchDims.y, launchDims.z, *stream>>>(offset, order, xPointer, xShapeInfoPointer, yPointer, yShapeInfoPointer, allocPointer);

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*stream));
}

void NativeOps::initializeDevicesAndFunctions() {
	int devCnt = 0;
	hipGetDeviceCount(&devCnt);
	deviceProperties = new hipDeviceProp_t[devCnt];
	for (int i = 0; i < devCnt; i++) {
		hipSetDevice(i);
		hipGetDeviceProperties(&deviceProperties[i], i);

		hipDeviceSetLimit(hipLimitStackSize, 10000);
		hipDeviceSetLimit(hipLimitMallocHeapSize , 10000);
	}

	hipSetDevice(0);

	hipFuncGetAttributes(&funcAttributes[0], reinterpret_cast<const void*>((void *))transformFloatIndexes);

	void (*transformFloatPointer1)(int opNum, float *dy,int *shapeInfo, int xRank, float *params, float *result,int *resultShapeInfo, int zRank, int *allocationPointer, float *reductionPointer) = transformFloat;
	hipFuncGetAttributes(&funcAttributes[1], reinterpret_cast<const void*>(transformFloatPointer1));

	void (*transformFloatPointer2)(int opNum, Nd4jIndex n, float *dy, int incy, float *params, float *result,int resultStride, int *allocationPointer, float *reductionPointer) = transformFloat;
	hipFuncGetAttributes(&funcAttributes[2], reinterpret_cast<const void*>(transformFloatPointer2));

	hipFuncGetAttributes(&funcAttributes[3], reinterpret_cast<const void*>((void *))summaryStatsReduceFloat);

	hipFuncGetAttributes(&funcAttributes[4], reinterpret_cast<const void*>((void *))scalarFloatIndexes);

	void (*scalarFloatPointer1)(int opNum, float dx,float *dy, int *shapeInfo, int xRank, float *params, float *result,int *resultShapeInfo, int zRank, int *allocPointer) = scalarFloat;
	hipFuncGetAttributes(&funcAttributes[5], reinterpret_cast<const void*>(scalarFloatPointer1));

	void (*scalarFloatPointer2)(int opNum, Nd4jIndex n,float dx, float *dy, int incy, float *params, float *result,int resultStride, int *allocPointer) = scalarFloat;
	hipFuncGetAttributes(&funcAttributes[6], reinterpret_cast<const void*>(scalarFloatPointer2));

	hipFuncGetAttributes(&funcAttributes[7], reinterpret_cast<const void*>(reduce3Float));

	hipFuncGetAttributes(&funcAttributes[8], reinterpret_cast<const void*>(reduceFloat));

	hipFuncGetAttributes(&funcAttributes[9], reinterpret_cast<const void*>(pairWiseTransformFloat));

	hipFuncGetAttributes(&funcAttributes[10], reinterpret_cast<const void*>(pairWiseTransformFloatIndex));

	hipFuncGetAttributes(&funcAttributes[11], reinterpret_cast<const void*>(pairWiseTransformStridedFloat));

	hipFuncGetAttributes(&funcAttributes[12], reinterpret_cast<const void*>(broadcastFloat));

	hipFuncGetAttributes(&funcAttributes[13], reinterpret_cast<const void*>(indexReduceFloat));

	///////////////////////////////////////// Doubles are separate, just in case of...

	hipFuncGetAttributes(&funcAttributes[14], reinterpret_cast<const void*>(transformDoubleIndexes));

	void (*transformDoublePointer1)(int opNum, double *dy, int *shapeInfo, int xRank, double *params, double *result,int *resultShapeInfo, int zRank, int *allocationPointer, double *reductionPointer) = transformDouble;
	hipFuncGetAttributes(&funcAttributes[15], reinterpret_cast<const void*>(transformDoublePointer1));

	void (*transformDoublePointer2)(int opNum, Nd4jIndex n, double *dy, int incy, double *params, double *result,int resultStride, int *allocationPointer, double *reductionPointer) = transformDouble;
	hipFuncGetAttributes(&funcAttributes[16], reinterpret_cast<const void*>(transformDoublePointer2));

	hipFuncGetAttributes(&funcAttributes[17], reinterpret_cast<const void*>(summaryStatsReduceDouble));

	hipFuncGetAttributes(&funcAttributes[18], reinterpret_cast<const void*>(scalarDoubleIndexes));

	void (*scalarDoublePointer1)(int opNum, double dx,double *dy, int *shapeInfo, int xRank, double *params, double *result,int *resultShapeInfo, int zRank, int *allocPointer) = scalarDouble;
	hipFuncGetAttributes(&funcAttributes[19], reinterpret_cast<const void*>(scalarDoublePointer1));


	void (*scalarDoublePointer2)(int opNum, Nd4jIndex n,double dx, double *dy, int incy, double *params, double *result,int resultStride, int *allocPointer) = scalarDouble;
	hipFuncGetAttributes(&funcAttributes[20], reinterpret_cast<const void*>(scalarDoublePointer2));

	hipFuncGetAttributes(&funcAttributes[21], reinterpret_cast<const void*>(reduce3Double));

	hipFuncGetAttributes(&funcAttributes[22], reinterpret_cast<const void*>(reduceDouble));

	hipFuncGetAttributes(&funcAttributes[23], reinterpret_cast<const void*>(pairWiseTransformDouble));

	hipFuncGetAttributes(&funcAttributes[24], reinterpret_cast<const void*>(pairWiseTransformDoubleIndex));

	hipFuncGetAttributes(&funcAttributes[25], reinterpret_cast<const void*>(pairWiseTransformStridedDouble));

	hipFuncGetAttributes(&funcAttributes[26], reinterpret_cast<const void*>(broadcastDouble));

	hipFuncGetAttributes(&funcAttributes[27], reinterpret_cast<const void*>(indexReduceDouble));
}


/**
 * This method acquires memory chunk of requested size on host side
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param flags optional parameter
 */
Nd4jPointer NativeOps::mallocHost(long memorySize, int flags) {
	Nd4jPointer pointer;
	hipError_t res = hipHostAlloc((void **)&pointer, memorySize, hipHostMallocMapped |hipHostMallocPortable );
	if (res != 0)
		pointer = 0L;
	return pointer;
}

/**
 * This method acquires memory chunk of requested size on specified device
 *
 * @param pointer pointer that'll be used for allocation
 * @param memorySize memory size, in bytes
 * @param ptrToDeviceId pointer to deviceId. For cuda that's just and int, for OpenCL that's pointer to device_id, etc
 * @param flags optional parameter
 */
Nd4jPointer NativeOps::mallocDevice(long memorySize, Nd4jPointer ptrToDeviceId, int flags) {
	Nd4jPointer pointer;
	hipError_t res = hipMalloc((void **)&pointer, memorySize);
	if (res != 0)
		pointer = 0L;
	return pointer;
}

/**
 * This method releases previously allocated host memory space
 *
 * @param pointer pointer that'll be freed
 */
Nd4jPointer NativeOps::freeHost(Nd4jPointer pointer) {
	hipError_t res = hipHostFree((void *) pointer);
	if (res != 0)
		pointer = 0L;
	return 1L;
}

/**
 * This method releases previously allocated memory space on device
 *
 * @param pointer pointer that'll be freed
 * @param ptrToDeviceId pointer to deviceId.
 */
Nd4jPointer NativeOps::freeDevice(Nd4jPointer pointer, Nd4jPointer ptrToDeviceId) {
	hipError_t res = hipFree((void *)pointer);
	if (res != 0)
		pointer = 0L;
	return 1L;
}


Nd4jPointer NativeOps::createContext() {
	return 0L;
}

Nd4jPointer NativeOps::createStream() {
	Nd4jPointer nativeStream = 0;
	hipError_t result = hipStreamCreate((hipStream_t *) &nativeStream);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return nativeStream;
}

Nd4jPointer NativeOps::createEvent() {
	Nd4jPointer nativeEvent= 0;
	hipError_t result = hipEventCreateWithFlags((hipEvent_t *) &nativeEvent, hipEventBlockingSync | hipEventDisableTiming);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return nativeEvent;
}

Nd4jPointer NativeOps::createBlasHandle() {
	Nd4jPointer nativeHandle= 0;
	hipblasStatus_t result = hipblasCreate((hipblasHandle_t *) &nativeHandle);
	if (result != 0)
		return 0L;
	else return nativeHandle;
}

Nd4jPointer NativeOps::registerEvent(Nd4jPointer event, Nd4jPointer stream) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&stream);

	hipError_t result = hipEventRecord(*pEvent, *pStream);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1;
}

Nd4jPointer NativeOps::setBlasStream(Nd4jPointer handle, Nd4jPointer stream) {
	hipblasHandle_t *pHandle = reinterpret_cast<hipblasHandle_t *>(&handle);
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&stream);

	hipblasStatus_t result = hipblasSetStream(*pHandle, *pStream);
	if (result != 0)
		return 0L;
	else return 1L;
}

Nd4jPointer NativeOps::setDevice(Nd4jPointer ptrToDeviceId) {
	int deviceId = (int) ptrToDeviceId;
	hipError_t result = hipSetDevice(deviceId);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1;
}

long NativeOps::getDeviceFreeMemory(Nd4jPointer ptrToDeviceId) {
	int device = (int) ptrToDeviceId;

	if (device >= 0) {
		setDevice(ptrToDeviceId);
	}
	size_t memFree = 0;
	size_t memTotal = 0;

	hipMemGetInfo(&memFree, &memTotal);

	return (long) memFree;
}

Nd4jPointer NativeOps::memcpy(Nd4jPointer dst, Nd4jPointer src, long size, int flags, Nd4jPointer reserved) {

	return memcpyAsync(dst, src, size, flags, reserved);
}

Nd4jPointer NativeOps::memcpyAsync(Nd4jPointer dst, Nd4jPointer src, long size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&reserved);

	hipMemcpyKind 	kind;

	if (debug)
		checkCudaErrors(hipStreamSynchronize(*pStream));

	switch (flags) {
		case 0: {
				kind = hipMemcpyHostToHost;
			}
			break;
		case 1: {
				kind = hipMemcpyHostToDevice;
			}
			break;
		case 2: {
				kind = hipMemcpyDeviceToHost;
			}
		case 3: {
			kind = hipMemcpyDeviceToDevice;
		}
			break;
	}

	hipError_t result = hipMemcpyAsync((void *) dst, (const void *) src, (size_t) size, kind, *pStream);
	checkCudaErrors(result);
	if (result != 0) {
		printf("Failed on [%lu] -> [%lu], size: [%i], direction: [%i]\n", src, dst, size, flags );
		return 0L;
	}
	else return 1;
}

Nd4jPointer NativeOps::memset(Nd4jPointer dst, int value, long size, int flags, Nd4jPointer reserved) {
	//hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&reserved);

	hipError_t result = hipMemset((void *) dst, value, (size_t) size);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1;
}

Nd4jPointer NativeOps::memsetAsync(Nd4jPointer dst, int value, long size, int flags, Nd4jPointer reserved) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&reserved);

	hipError_t result = hipMemsetAsync((void *) dst, value, (size_t) size, *pStream);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1;
}

Nd4jPointer NativeOps::destroyEvent(Nd4jPointer event) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);
	hipError_t result = hipEventDestroy(*pEvent);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1;
}

Nd4jPointer NativeOps::streamSynchronize(Nd4jPointer stream) {
	hipStream_t *pStream = reinterpret_cast<hipStream_t *>(&stream);

	hipError_t result = hipStreamSynchronize(*pStream);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1L;
}

Nd4jPointer NativeOps::eventSynchronize(Nd4jPointer event) {
	hipEvent_t *pEvent = reinterpret_cast<hipEvent_t *>(&event);

	hipError_t result = hipEventSynchronize(*pEvent);
	checkCudaErrors(result);
	if (result != 0)
		return 0L;
	else return 1L;
}

Nd4jPointer NativeOps::getAvailableDevices() {
	int devCnt = 0;
	hipGetDeviceCount(&devCnt);
	return (Nd4jPointer) devCnt;
}

void NativeOps::enableDebugMode(bool reallyEnable) {
	debug = reallyEnable;
}

void NativeOps::setGridLimit(int gridSize) {
	if (gridSize > 1024)
		gridSize = 1024;
	if (gridSize < 1)
		gridSize = 1;
	blockLimit = gridSize;
}


int NativeOps::ompGetNumThreads() {
	return maxThreads;
}

void NativeOps::setOmpNumThreads(int threads) {
	if (threads > 1024)
		threads = 1024;
	if (threads < 32)
		threads = 32;
	maxThreads = threads;
}
