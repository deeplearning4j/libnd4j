#include "hip/hip_runtime.h"



#include <op_boilerplate.h>
#include <helpers/TAD.h>
#include <types/float16.h>
#include <loops/scalar.h>
#include <loops/pairwise_transform.h>
#include "../grid.h"



#define GRID_WIDTH 19 // number of pointers within single grid row

/*
template <typename T>
__device__ inline static void metaPredicateReduceGeneric(const int opTypeA, const int opNumA, const int opTypeB, const int opNumB,
                                                         T *dx, int *xShapeInfo, T *dy, int *yShapeInfo, T *dz, int *zShapeInfo, int *dimension, int dimensionLength, int *tadShapeInfo, Nd4jIndex *tadOffsets, T *reductionBuffer, T *extraA, T *extraB, T scalarA, T scalarB, bool scalarReturned) {

    __shared__ UnifiedSharedMemory *manager;
    __shared__ Nd4jPointer params[2];
    __shared__ T *paramsPtr;
    if (threadIdx.x == 0) {
        if (opTypeA == 0) {
            params[0] = (Nd4jPointer *) &scalarA;
        }
        else params[0] = (Nd4jPointer *) extraA;

        if (opTypeB == 0) {
            params[1] = (Nd4jPointer *) &scalarB;
        }
        else params[1] = (Nd4jPointer *) extraB;

        paramsPtr = (T *) params;

        extern __shared__ unsigned char shmem[];
        manager = new(shmem) UnifiedSharedMemory((int *) shmem);
        manager->init(sizeof(UnifiedSharedMemory), 0, sizeof(functions::reduce::ReduceFunction<T>), sizeof(shape::TAD), shape::rank(xShapeInfo));
    }
    __syncthreads();

    // this op can be used only for reduce calls now

    if (opTypeA == 0) { // scalar

        //   DISPATCH_METAOP(functions::reduce::ReduceFunction<T>::template transformCuda6D, PARAMS(dx, xShapeInfo, paramsPtr, dz, zShapeInfo,  dimension, dimensionLength, reductionBuffer, manager, tadShapeInfo, tadOffsets), ReduceMetaOp, OPS_A(SCALAR_OPS), OPS_B(REDUCE_OPS));
    } else if (opTypeA == 1) { // transform

    } else {
        if (threadIdx.x == 0 && blockIdx.x == 0)
            printf("Unknown opTypeA: [%i]\n", opTypeA);
    }
}
*/
template <typename T>
__device__ inline static void metaPredicateShapeGeneric(const int opTypeA, const int opNumA, const int opTypeB, const int opNumB,
                                                        long N, T *dx, int *xShapeInfo, T *dy, int *yShapeInfo, T *dz, int *zShapeInfo, T *extraA, T *extraB, T scalarA, T scalarB) {
    __shared__ Nd4jPointer params[2];
    __shared__ T *paramsPtr;
    if (threadIdx.x == 0) {
        if (opTypeA == 0) {
            params[0] = (Nd4jPointer *) &scalarA;
        }
        else params[0] = (Nd4jPointer *) extraA;

        if (opTypeB == 0) {
            params[1] = (Nd4jPointer *) &scalarB;
        }
        else params[1] = (Nd4jPointer *) extraB;

        paramsPtr = (T *) params;
    }
    __syncthreads();

    if (opTypeA == 2) {
        if (opTypeB == 0) {
            //    DISPATCH_METAOP(functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda, PARAMS(dx, xShapeInfo, dy, yShapeInfo, dz, zShapeInfo, paramsPtr, nullptr, nullptr, nullptr), InvertedMetaOp, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
            //  functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda<simdOps::InvertedMetaOp<T, simdOps::Copy<T>, simdOps::Multiply<T>>>(dx, xShapeInfo, dy, yShapeInfo, dz, zShapeInfo, paramsPtr, nullptr, nullptr, nullptr);
        }
    }
}

template<typename T, typename OpClass>
__device__ static inline void invertedMetaPairwiseShapedGeneric(const int opTypeA, const int opTypeB, long N, T *dx, int *xShapeInfo, T *dy, int *yShapeInfo, T *dz, int *zShapeInfo, T *extraA, T *extraB, T scalarA, T scalarB) {
    __shared__ Nd4jPointer params[2];
    __shared__ T *paramsPtr;
    if (threadIdx.x == 0) {
        if (opTypeA == 0) {
            params[0] = (Nd4jPointer *) &scalarA;
        }
        else params[0] = (Nd4jPointer *) extraA;

        if (opTypeB == 0) {
            params[1] = (Nd4jPointer *) &scalarB;
        }
        else params[1] = (Nd4jPointer *) extraB;

        paramsPtr = (T *) params;
    }
    __syncthreads();

    functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda<OpClass>(dx, xShapeInfo, dy, yShapeInfo, dz, zShapeInfo, paramsPtr, nullptr, nullptr, nullptr);
};



// kernels set for pairwise + scalar based on shape
DISPATCH_KERNEL_META(invertedMetaPairwiseShaped_Pairwise_Scalar_, invertedMetaPairwiseShapedGeneric, float, simdOps::InvertedMetaOp, INPUT(const int opTypeA, const int opTypeB, long N, float *dx, int *xShapeInfo, float *dy, int *yShapeInfo, float *dz, int *zShapeInfo, float *extraA, float *extraB, float scalarA, float scalarB), PARAMS(opTypeA, opTypeB, N, dx, xShapeInfo, dy, yShapeInfo, dz, zShapeInfo, extraA, extraB, scalarA, scalarB),  OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS))
DISPATCH_KERNEL_META(invertedMetaPairwiseShaped_Pairwise_Scalar_, invertedMetaPairwiseShapedGeneric, double, simdOps::InvertedMetaOp, INPUT(const int opTypeA, const int opTypeB, long N, double *dx, int *xShapeInfo, double *dy, int *yShapeInfo, double *dz, int *zShapeInfo, double *extraA, double *extraB, double scalarA, double scalarB), PARAMS(opTypeA, opTypeB, N, dx, xShapeInfo, dy, yShapeInfo, dz, zShapeInfo, extraA, extraB, scalarA, scalarB),  OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS))
DISPATCH_KERNEL_META(invertedMetaPairwiseShaped_Pairwise_Scalar_, invertedMetaPairwiseShapedGeneric, float16, simdOps::InvertedMetaOp, INPUT(const int opTypeA, const int opTypeB, long N, float16 *dx, int *xShapeInfo, float16 *dy, int *yShapeInfo, float16 *dz, int *zShapeInfo, float16 *extraA, float16 *extraB, float16 scalarA, float16 scalarB), PARAMS(opTypeA, opTypeB, N, dx, xShapeInfo, dy, yShapeInfo, dz, zShapeInfo, extraA, extraB, scalarA, scalarB),  OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS))




namespace functions {
    namespace grid {

        template <>
        void GRID<float>::execMetaPredicateShaped(hipStream_t * stream, Nd4jPointer *extras, const int opTypeA, const int opNumA, const int opTypeB, const int opNumB, long N, float *dx, int *xShapeInfo, float *dy, int *yShapeInfo, float *dz, int *zShapeInfo, float *extraA, float *extraB, float scalarA, float scalarB) {
            if (opTypeA == 2) {
                if (opTypeB == 0) {
                    DISPATCH_METAOP(invertedMetaPairwiseShaped_Pairwise_Scalar, PARAMS(opTypeA, opTypeB, N, dx, xShapeInfo, dy, yShapeInfo, dz, zShapeInfo, extraA, extraB, scalarA, scalarB), float, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
                }
            }
        }

        template <>
        void GRID<float16>::execMetaPredicateShaped(hipStream_t * stream, Nd4jPointer *extras, const int opTypeA, const int opNumA, const int opTypeB, const int opNumB, long N, float16 *dx, int *xShapeInfo, float16 *dy, int *yShapeInfo, float16 *dz, int *zShapeInfo, float16 *extraA, float16 *extraB, float16 scalarA, float16 scalarB) {
            if (opTypeA == 2) {
                if (opTypeB == 0) {
                    DISPATCH_METAOP(invertedMetaPairwiseShaped_Pairwise_Scalar, PARAMS(opTypeA, opTypeB, N, dx, xShapeInfo, dy, yShapeInfo, dz, zShapeInfo, extraA, extraB, scalarA, scalarB), float16, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
                }
            }
        }

        template <>
        void GRID<double>::execMetaPredicateShaped(hipStream_t * stream, Nd4jPointer *extras, const int opTypeA, const int opNumA, const int opTypeB, const int opNumB, long N, double *dx, int *xShapeInfo, double *dy, int *yShapeInfo, double *dz, int *zShapeInfo, double *extraA, double *extraB, double scalarA, double scalarB) {
            if (opTypeA == 2) {
                if (opTypeB == 0) {
                    DISPATCH_METAOP(invertedMetaPairwiseShaped_Pairwise_Scalar, PARAMS(opTypeA, opTypeB, N, dx, xShapeInfo, dy, yShapeInfo, dz, zShapeInfo, extraA, extraB, scalarA, scalarB), double, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
                }
            }
        }


        //template class GRID<float>;
        //template class GRID<float16>;
        //template class GRID<double>;
    }
}