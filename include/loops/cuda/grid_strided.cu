#include "hip/hip_runtime.h"



#include <op_boilerplate.h>
#include <helpers/TAD.h>
#include <loops/scalar.h>
#include <loops/pairwise_transform.h>
#include "../grid.h"
#include <types/float16.h>


#define GRID_WIDTH 19 // number of pointers within single grid row

/*
template <typename T>
__device__ inline static void metaPredicateReduceGeneric(const int opTypeA, const int opNumA, const int opTypeB, const int opNumB,
                                                         T *dx, int *xShapeInfo, T *dy, int *yShapeInfo, T *dz, int *zShapeInfo, int *dimension, int dimensionLength, int *tadShapeInfo, Nd4jIndex *tadOffsets, T *reductionBuffer, T *extraA, T *extraB, T scalarA, T scalarB, bool scalarReturned) {

    __shared__ UnifiedSharedMemory *manager;
    __shared__ Nd4jPointer params[2];
    __shared__ T *paramsPtr;
    if (threadIdx.x == 0) {
        if (opTypeA == 0) {
            params[0] = (Nd4jPointer *) &scalarA;
        }
        else params[0] = (Nd4jPointer *) extraA;

        if (opTypeB == 0) {
            params[1] = (Nd4jPointer *) &scalarB;
        }
        else params[1] = (Nd4jPointer *) extraB;

        paramsPtr = (T *) params;

        extern __shared__ unsigned char shmem[];
        manager = new(shmem) UnifiedSharedMemory((int *) shmem);
        manager->init(sizeof(UnifiedSharedMemory), 0, sizeof(functions::reduce::ReduceFunction<T>), sizeof(shape::TAD), shape::rank(xShapeInfo));
    }
    __syncthreads();

    // this op can be used only for reduce calls now

    if (opTypeA == 0) { // scalar

        //   DISPATCH_METAOP(functions::reduce::ReduceFunction<T>::template transformCuda6D, PARAMS(dx, xShapeInfo, paramsPtr, dz, zShapeInfo,  dimension, dimensionLength, reductionBuffer, manager, tadShapeInfo, tadOffsets), ReduceMetaOp, OPS_A(SCALAR_OPS), OPS_B(REDUCE_OPS));
    } else if (opTypeA == 1) { // transform

    } else {
        if (threadIdx.x == 0 && blockIdx.x == 0)
            printf("Unknown opTypeA: [%i]\n", opTypeA);
    }
}
*/

template <typename T>
__device__ inline static void metaPredicateStridedGeneric(const int opTypeA, const int opNumA, const int opTypeB, const int opNumB,
                                                          long N, T *dx, int xStride, T *dy, int yStride, T *dz, int zStride, T *extraA, T *extraB, T scalarA, T scalarB
) {
    __shared__ Nd4jPointer params[2];
    __shared__ T *paramsPtr;
    if (threadIdx.x == 0) {
        if (opTypeA == 0) {
            params[0] = (Nd4jPointer *) &scalarA;
        }
        else params[0] = (Nd4jPointer *) extraA;

        if (opTypeB == 0) {
            params[1] = (Nd4jPointer *) &scalarB;
        }
        else params[1] = (Nd4jPointer *) extraB;

        paramsPtr = (T *) params;
    }
    __syncthreads();
#ifdef __EXPERIMENTAL__
    if (opTypeB == 0) { // SCALAR
        if (opTypeA == 0) {
            // double scalar
            DISPATCH_METAOP(functions::transform::Transform<T>::template transformCuda, PARAMS(N, dx, xStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), MetaOp, OPS_A(SCALAR_OPS), OPS_B(SCALAR_OPS));
        } else if (opTypeA == 1) {
            // transform
            DISPATCH_METAOP(functions::transform::Transform<T>::template transformCuda, PARAMS(N, dx, xStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), MetaOp, OPS_A(TRANSFORM_OPS), OPS_B(SCALAR_OPS));
        } else if (opTypeA == 2) {
            // pwt
            // this is the most important thing here: its Dup() + Scalar
            DISPATCH_METAOP(functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda, PARAMS(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), InvertedMetaOp, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
        }
    } else if (opTypeB == 1) { // TRANSFORM
        if (opTypeA == 0) {
            DISPATCH_METAOP(functions::transform::Transform<T>::template transformCuda, PARAMS(N, dx, xStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), MetaOp, OPS_A(SCALAR_OPS), OPS_B(TRANSFORM_OPS));
        }
    } else if (opTypeB == 2) { // PWT
        if (opTypeA == 0) { // SCALAR

            DISPATCH_METAOP(functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda, PARAMS(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), MetaOp, OPS_A(SCALAR_OPS), OPS_B(PAIRWISE_TRANSFORM_OPS));
        } else if (opTypeA == 1) { // TRANSFORM

            DISPATCH_METAOP(functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda, PARAMS(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), MetaOp, OPS_A(TRANSFORM_OPS), OPS_B(PAIRWISE_TRANSFORM_OPS));
        } else if (opTypeA == 2) {

        }
    } else {
        if (threadIdx.x == 0 && blockIdx.x)
            printf("Unknown opTypeB: [%i]\n", opTypeB);
    }
#else
    if (opTypeA == 2) {
        if (opTypeB == 0) {
            //      DISPATCH_METAOP(functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda, PARAMS(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), InvertedMetaOp, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
            //      functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda<simdOps::InvertedMetaOp<T, simdOps::Copy<T>, simdOps::Multiply<T>>>(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr);
        }
    }
#endif
}

template<typename T, typename OpClass>
__device__ static inline void invertedMetaPairwiseStridedGeneric(const int opTypeA, const int opTypeB, long N, T *dx, int xStride, T *dy, int yStride, T *dz, int zStride, T *extraA, T *extraB, T scalarA, T scalarB) {
    __shared__ Nd4jPointer params[2];
    __shared__ T *paramsPtr;
    if (threadIdx.x == 0) {
        if (opTypeA == 0) {
            params[0] = (Nd4jPointer *) &scalarA;
        }
        else params[0] = (Nd4jPointer *) extraA;

        if (opTypeB == 0) {
            params[1] = (Nd4jPointer *) &scalarB;
        }
        else params[1] = (Nd4jPointer *) extraB;

        paramsPtr = (T *) params;
    }
    __syncthreads();

    functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda<OpClass>(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr);
};


// kernels set for pairwise + scalar based on stride                                                                                         const int opTypeA, const int opTypeB, long N, T *dx, int xStride, T *dy, int yStride, T *dz, int zStride, T *extraA, T *extraB, T scalarA, T scalarB
DISPATCH_KERNEL_META(invertedMetaPairwiseStrided_Pairwise_Scalar_, invertedMetaPairwiseStridedGeneric, float, simdOps::InvertedMetaOp, INPUT(const int opTypeA, const int opTypeB, long N, float *dx, int xStride, float *dy, int yStride, float *dz, int zStride, float *extraA, float *extraB, float scalarA, float scalarB), PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB),  OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS))
DISPATCH_KERNEL_META(invertedMetaPairwiseStrided_Pairwise_Scalar_, invertedMetaPairwiseStridedGeneric, double, simdOps::InvertedMetaOp, INPUT(const int opTypeA, const int opTypeB, long N, double *dx, int xStride, double *dy, int yStride, double *dz, int zStride, double *extraA, double *extraB, double scalarA, double scalarB), PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB),  OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS))
DISPATCH_KERNEL_META(invertedMetaPairwiseStrided_Pairwise_Scalar_, invertedMetaPairwiseStridedGeneric, float16, simdOps::InvertedMetaOp, INPUT(const int opTypeA, const int opTypeB, long N, float16 *dx, int xStride, float16 *dy, int yStride, float16 *dz, int zStride, float16 *extraA, float16 *extraB, float16 scalarA, float16 scalarB), PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB),  OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS))



namespace functions {
    namespace grid {

        template <>
        void GRID<float>::execMetaPredicateStrided(hipStream_t * stream, Nd4jPointer *extras, const int opTypeA, const int opNumA, const int opTypeB, const int opNumB, long N, float *dx, int xStride, float *dy, int yStride, float *dz, int zStride, float *extraA, float *extraB, float scalarA, float scalarB) {
            if (opTypeA == 2) {
                if (opTypeB == 0) {
                    DISPATCH_METAOP(invertedMetaPairwiseStrided_Pairwise_Scalar, PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB), float, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
                }
            }
        }

        template <>
        void GRID<float16>::execMetaPredicateStrided(hipStream_t * stream, Nd4jPointer *extras, const int opTypeA, const int opNumA, const int opTypeB, const int opNumB, long N, float16 *dx, int xStride, float16 *dy, int yStride, float16 *dz, int zStride, float16 *extraA, float16 *extraB, float16 scalarA, float16 scalarB) {
            if (opTypeA == 2) {
                if (opTypeB == 0) {
                    DISPATCH_METAOP(invertedMetaPairwiseStrided_Pairwise_Scalar, PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB), float16, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
                }
            }
        }

        template <>
        void GRID<double>::execMetaPredicateStrided(hipStream_t * stream, Nd4jPointer *extras, const int opTypeA, const int opNumA, const int opTypeB, const int opNumB, long N, double *dx, int xStride, double *dy, int yStride, double *dz, int zStride, double *extraA, double *extraB, double scalarA, double scalarB) {
            if (opTypeA == 2) {
                if (opTypeB == 0) {
                    DISPATCH_METAOP(invertedMetaPairwiseStrided_Pairwise_Scalar, PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB), double, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
                }
            }
        }

        //template class GRID<float>;
        //template class GRID<float16>;
        //template class GRID<double>;
    }
}