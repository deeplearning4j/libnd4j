#include "hip/hip_runtime.h"



#include <op_boilerplate.h>
#include <helpers/TAD.h>
#include "../grid_strided.h"
#include <types/float16.h>


#define GRID_WIDTH 19 // number of pointers within single grid row

#include <ops/ops.h>
#include "../legacy_ops.h"

template <typename T>
__device__ inline static void metaPredicateStridedGeneric(const int opTypeA, const int opNumA, const int opTypeB, const int opNumB,
                                                          long N, T *dx, int xStride, T *dy, int yStride, T *dz, int zStride, T *extraA, T *extraB, T scalarA, T scalarB
) {
    __shared__ Nd4jPointer params[2];
    __shared__ T *paramsPtr;
    if (threadIdx.x == 0) {
        if (opTypeA == 0) {
            params[0] = (Nd4jPointer *) &scalarA;
        }
        else params[0] = (Nd4jPointer *) extraA;

        if (opTypeB == 0) {
            params[1] = (Nd4jPointer *) &scalarB;
        }
        else params[1] = (Nd4jPointer *) extraB;

        paramsPtr = (T *) params;
    }
    __syncthreads();
#ifdef __EXPERIMENTAL__
    if (opTypeB == 0) { // SCALAR
        if (opTypeA == 0) {
            // double scalar
            DISPATCH_METAOP(functions::transform::Transform<T>::template transformCuda, PARAMS(N, dx, xStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), MetaOp, OPS_A(SCALAR_OPS), OPS_B(SCALAR_OPS));
        } else if (opTypeA == 1) {
            // transform
            DISPATCH_METAOP(functions::transform::Transform<T>::template transformCuda, PARAMS(N, dx, xStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), MetaOp, OPS_A(TRANSFORM_OPS), OPS_B(SCALAR_OPS));
        } else if (opTypeA == 2) {
            // pwt
            // this is the most important thing here: its Dup() + Scalar
            DISPATCH_METAOP(functions::grid::GRID<T>::template transformCuda, PARAMS(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), InvertedMetaOp, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
        }
    } else if (opTypeB == 1) { // TRANSFORM
        if (opTypeA == 0) {
            DISPATCH_METAOP(functions::transform::Transform<T>::template transformCuda, PARAMS(N, dx, xStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), MetaOp, OPS_A(SCALAR_OPS), OPS_B(TRANSFORM_OPS));
        }
    } else if (opTypeB == 2) { // PWT
        if (opTypeA == 0) { // SCALAR

            DISPATCH_METAOP(functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda, PARAMS(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), MetaOp, OPS_A(SCALAR_OPS), OPS_B(PAIRWISE_TRANSFORM_OPS));
        } else if (opTypeA == 1) { // TRANSFORM

            DISPATCH_METAOP(functions::grid::GRID<T>::template transformCuda, PARAMS(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), MetaOp, OPS_A(TRANSFORM_OPS), OPS_B(PAIRWISE_TRANSFORM_OPS));
        } else if (opTypeA == 2) {

        }
    } else {
        if (threadIdx.x == 0 && blockIdx.x)
            printf("Unknown opTypeB: [%i]\n", opTypeB);
    }
#else
    if (opTypeA == 2) {
        if (opTypeB == 0) {
            //      DISPATCH_METAOP(functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda, PARAMS(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr), InvertedMetaOp, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
            //      functions::pairwise_transforms::PairWiseTransform<T>::template transformCuda<simdOps::InvertedMetaOp<T, simdOps::Copy<T>, simdOps::Multiply<T>>>(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr);
        }
    }
#endif
}

template<typename T, typename OpClass>
__device__ static inline void invertedMetaPairwiseStridedGeneric(const int opTypeA, const int opTypeB, long N, T *dx, int xStride, T *dy, int yStride, T *dz, int zStride, T *extraA, T *extraB, T scalarA, T scalarB) {
    __shared__ Nd4jPointer params[2];
    __shared__ T *paramsPtr;
    if (threadIdx.x == 0) {
        if (opTypeA == 0) {
            params[0] = (Nd4jPointer *) &scalarA;
        }
        else params[0] = (Nd4jPointer *) extraA;

        if (opTypeB == 0) {
            params[1] = (Nd4jPointer *) &scalarB;
        }
        else params[1] = (Nd4jPointer *) extraB;

        paramsPtr = (T *) params;
    }
    __syncthreads();

    functions::grid::GRIDStrided<T>::template transformCuda<OpClass>(N, dx, dy, xStride, yStride, paramsPtr, dz, zStride, nullptr, nullptr, nullptr);
};

#ifndef __CLION_IDE__
// kernels set for pairwise + scalar based on stride                                                                                         const int opTypeA, const int opTypeB, long N, T *dx, int xStride, T *dy, int yStride, T *dz, int zStride, T *extraA, T *extraB, T scalarA, T scalarB
DISPATCH_KERNEL_META(invertedMetaPairwiseStrided_Pairwise_Scalar_, invertedMetaPairwiseStridedGeneric, float, simdOps::InvertedMetaOp, INPUT(const int opTypeA, const int opTypeB, long N, float *dx, int xStride, float *dy, int yStride, float *dz, int zStride, float *extraA, float *extraB, float scalarA, float scalarB), PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB),  OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS))
DISPATCH_KERNEL_META(invertedMetaPairwiseStrided_Pairwise_Scalar_, invertedMetaPairwiseStridedGeneric, double, simdOps::InvertedMetaOp, INPUT(const int opTypeA, const int opTypeB, long N, double *dx, int xStride, double *dy, int yStride, double *dz, int zStride, double *extraA, double *extraB, double scalarA, double scalarB), PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB),  OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS))
DISPATCH_KERNEL_META(invertedMetaPairwiseStrided_Pairwise_Scalar_, invertedMetaPairwiseStridedGeneric, float16, simdOps::InvertedMetaOp, INPUT(const int opTypeA, const int opTypeB, long N, float16 *dx, int xStride, float16 *dy, int yStride, float16 *dz, int zStride, float16 *extraA, float16 *extraB, float16 scalarA, float16 scalarB), PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB),  OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS))
#endif


namespace functions {
    namespace grid {

        template<typename T>
        template<typename OpType>
        __device__ void GRIDStrided<T>::transformCuda(Nd4jIndex n, T *dx, T *dy, int incx, int incy, T *params, T *result, int incz,int *allocationPointer, UnifiedSharedMemory *manager,int *tadOnlyShapeInfo) {
            int tid = blockIdx.x * blockDim.x + threadIdx.x;

            if (incx == incy && incy == incz && incx == 1) {
                for (Nd4jIndex i = tid; i < n; i += gridDim.x * blockDim.x) {
                    result[i] = OpType::op(dx[i], dy[i], params);
                }
            } else {
                for (Nd4jIndex i = tid; i < n; i += gridDim.x * blockDim.x) {
                    result[i * incz] = OpType::op(dx[i * incx], dy[i * incy], params);
                }
            }
        }


        template <>
        void GRIDStrided<float>::execMetaPredicateStrided(hipStream_t * stream, Nd4jPointer *extras, const int opTypeA, const int opNumA, const int opTypeB, const int opNumB, long N, float *dx, int xStride, float *dy, int yStride, float *dz, int zStride, float *extraA, float *extraB, float scalarA, float scalarB) {
            if (opTypeA == 2) {
                if (opTypeB == 0) {
                    DISPATCH_METAOP(invertedMetaPairwiseStrided_Pairwise_Scalar, PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB), float, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
                }
            }
        }

        template <>
        void GRIDStrided<float16>::execMetaPredicateStrided(hipStream_t * stream, Nd4jPointer *extras, const int opTypeA, const int opNumA, const int opTypeB, const int opNumB, long N, float16 *dx, int xStride, float16 *dy, int yStride, float16 *dz, int zStride, float16 *extraA, float16 *extraB, float16 scalarA, float16 scalarB) {
            if (opTypeA == 2) {
                if (opTypeB == 0) {
                    DISPATCH_METAOP(invertedMetaPairwiseStrided_Pairwise_Scalar, PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB), float16, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
                }
            }
        }

        template <>
        void GRIDStrided<double>::execMetaPredicateStrided(hipStream_t * stream, Nd4jPointer *extras, const int opTypeA, const int opNumA, const int opTypeB, const int opNumB, long N, double *dx, int xStride, double *dy, int yStride, double *dz, int zStride, double *extraA, double *extraB, double scalarA, double scalarB) {
            if (opTypeA == 2) {
                if (opTypeB == 0) {
                    DISPATCH_METAOP(invertedMetaPairwiseStrided_Pairwise_Scalar, PARAMS(opTypeA, opTypeB, N, dx, xStride, dy, yStride, dz, zStride, extraA, extraB, scalarA, scalarB), double, OPS_A(PAIRWISE_TRANSFORM_OPS), OPS_B(SCALAR_OPS));
                }
            }
        }

        //template class GRID<float>;
        //template class GRID<float16>;
        //template class GRID<double>;
    }
}